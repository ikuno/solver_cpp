#include "hip/hip_runtime.h"
#include "cudaFunction.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipsparse.h"
#include <hip/hip_runtime_api.h>
#ifdef _OPENMP
#include <omp.h>
#endif

#include "color.hpp"

__device__ __inline__ double shfl_xor(double value, int const lane)
{
  return __hiloint2double(__shfl_xor(__double2hiint(value), lane),
      __shfl_xor(__double2loint(value), lane)); 
}

__global__ void kernel_dot (const int N, const double *__restrict__ a, const double *__restrict__ b, double *c)
{
  extern __shared__ double cache[];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;

  double temp = 0;
  while (tid < N) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cacheIndex] = temp;
  __syncthreads ();

  int i = blockDim.x / 2;
  while (i != 0) {
    if (cacheIndex < i) {
      cache[cacheIndex] += cache[cacheIndex+i];
    }
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0) {
    c[blockIdx.x] = cache[0];
  }
}

__global__ void kernel_dot (const int N, const double *__restrict__ a, const int aindex, const int asize, const double *__restrict__ b, double *c, const int cindex, const int csize)
{
  extern __shared__ double cache[];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;

  double temp = 0;
  while (tid < N) {
    temp += a[aindex * asize + tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cacheIndex] = temp;
  __syncthreads ();

  int i = blockDim.x / 2;
  while (i != 0) {
    if (cacheIndex < i) {
      cache[cacheIndex] += cache[cacheIndex+i];
    }
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0) {
    c[cindex * csize + blockIdx.x] = cache[0];
  }
}

/* kernel_MtxVec_mult_old_1<<<BlockPerGrid, ThreadPerBlock>>>(size, val, col, ptr, D_in, D_out); */
__global__ void kernel_MtxVec_mult_old_1(unsigned long int n, double *val, int *col, int *ptr, double *b, double *c){
  long row=blockDim.x * blockIdx.x + threadIdx.x;
  long int i;
  if(row<n){
    double tmp=0.0;
    long int row_start=ptr[row];
    long int row_end=ptr[row+1];
    for(i=row_start;i<row_end;i++){
      tmp+=val[i]*b[col[i]];
    }
    c[row]=tmp;
  }
}

  /* kernel_MtxVec_mult_old_2<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(this->size, val, col, ptr, cu_d1, cu_d2); */
__global__ void kernel_MtxVec_mult_old_2(unsigned long int n, const double *val, const int *col, const int *ptr, const double *__restrict__ b, double *c){
  extern __shared__ volatile double vals[];

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    double sum = 0.0;
    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      sum += val[jj] * b[col[jj]];
    }

    vals[threadIdx.x] = sum;
    vals[threadIdx.x] = sum = sum + vals[threadIdx.x + 16];
    vals[threadIdx.x] = sum = sum + vals[threadIdx.x + 8];
    vals[threadIdx.x] = sum = sum + vals[threadIdx.x + 4];
    vals[threadIdx.x] = sum = sum + vals[threadIdx.x + 2];
    sum = sum + vals[threadIdx.x+1];

    if(lane == 0){
      c[row] = sum;
    }
  }
}

/* kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d2); */
__global__ void kernel_MtxVec_mult(unsigned long int n, const double *val, const int *col, const int *ptr, const double *__restrict__ b, double *c){

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    double sum = 0.0;
    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      sum += val[jj] * b[col[jj]];
    }

    sum += shfl_xor(sum, 16);
    sum += shfl_xor(sum, 8);
    sum += shfl_xor(sum, 4);
    sum += shfl_xor(sum, 2);
    sum += shfl_xor(sum, 1);


    if(lane == 0){
      c[row] = sum;
    }
  }
}

__global__ void kernel_MtxVec_mult(unsigned long int n, const double *val, const int *col, const int *ptr, const double *__restrict__ b, double *c, const int cindex, const int csize){

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    double sum = 0.0;
    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      sum += val[jj] * b[col[jj]];
    }

    sum += shfl_xor(sum, 16);
    sum += shfl_xor(sum, 8);
    sum += shfl_xor(sum, 4);
    sum += shfl_xor(sum, 2);
    sum += shfl_xor(sum, 1);


    if(lane == 0){
      c[cindex * csize + row] = sum;
    }
  }
}

__global__ void kernel_MtxVec_mult(int n, const double *val, const int *col, const int *ptr, const double *__restrict__ b, const int bindex, const int bsize, double *c, const int cindex, const int csize){

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    double sum = 0.0;
    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      sum += val[jj] * b[bindex * bsize + col[jj]];
    }

    sum += shfl_xor(sum, 16);
    sum += shfl_xor(sum, 8);
    sum += shfl_xor(sum, 4);
    sum += shfl_xor(sum, 2);
    sum += shfl_xor(sum, 1);


    if(lane == 0){
      c[cindex * csize + row] = sum;
    }
  }
}

//----------------------------------------------------------------------------------------------

cuda::cuda(times *t){

  this->time = t;

  this->cu_d1 = NULL;
  this->cu_d2 = NULL;

  this->cu_d3 = NULL;
  this->cu_h1 = NULL;

  this->cu_d4 = NULL;
  this->cu_d5 = NULL;
  this->cu_d6 = NULL;
  this->cu_d7 = NULL;
  this->cu_d8 = NULL;
  
  this->cu_d9 = NULL;

  this->cu_h2 = NULL;
  this->cu_h3 = NULL;
  this->cu_h4 = NULL;

  this->cu_h5 = NULL;

  this->cu_d10 = NULL;
  this->cu_d11 = NULL;
  
  this->cu_h6 = NULL;
  this->cu_h7 = NULL;

  this->cu_d100 = NULL;
  this->cu_d101 = NULL;
  this->cu_d200 = NULL;
  this->cu_d201 = NULL;


  this->cu_d1_1 = NULL;
  this->cu_d1_2 = NULL;
  this->cu_d2_1 = NULL;
  this->cu_d2_2 = NULL;

  this->cu_d4_1 = NULL;
  this->cu_d4_2 = NULL;
  this->cu_d5_1 = NULL;
  this->cu_d5_2 = NULL;

}

cuda::cuda(times *t, unsigned long int size, unsigned long int size1, unsigned long int size2) : cuda::cuda(t){

  if((size1 == size2) && size1 == 0){
    isMulti = false;
  }else{
    isMulti = true;
  }
  
  if(isMulti){
    this->size1 = size1;
    this->size2 = size2;
    this->size = size;
  }else{
    this->size = size;
  }

  int tmp = ceil((double)this->size/(double)128);
  int tmp1 = ceil((double)this->size1/(double)128);
  int tmp2 = ceil((double)this->size2/(double)128);
  if(isMulti){
    this->cu_d1_1 = d_Malloc(this->size, 0);
    this->cu_d2_1 = d_Malloc(this->size1, 0);

    this->cu_d1_2 = d_Malloc(this->size, 1);
    this->cu_d2_2 = d_Malloc(this->size2, 1);
  }else{
    this->cu_d1 = d_Malloc(this->size);
    this->cu_d2 = d_Malloc(this->size);
  }

  if(isMulti){
    //dot not available in multi gpu now
  }else{
    this->cu_d3 = d_Malloc(tmp);
    this->cu_h1 = d_MallocHost(tmp);
  }

  this->cu_h8 = d_MallocHost(tmp * (1000+1));
}

cuda::cuda(times *t, unsigned long int size, int k, unsigned long int size1, unsigned long int size2) : cuda::cuda(t, size, size1, size2){
  this->k = k;
  int tmp = ceil((double)this->size/(double)128);

  if(isMulti){
    this->cu_d4_1 = d_Malloc(this->size * (2*this->k + 1), 0);
    this->cu_d4_2 = d_Malloc(this->size * (2*this->k + 1), 1);

    this->cu_d5_1 = d_Malloc(this->size * (2*this->k + 2), 0);
    this->cu_d5_2 = d_Malloc(this->size * (2*this->k + 2), 1);
  }else{
    this->cu_d4 = d_Malloc(this->size * (2*this->k + 1));
    this->cu_d5 = d_Malloc(this->size * (2*this->k + 2));
  }

  if(isMulti){
  }else{
    this->cu_d6 = d_Malloc(tmp * (2*this->k));
    this->cu_d7 = d_Malloc(tmp * (2*this->k + 1));
    this->cu_d8 = d_Malloc(tmp * (2*this->k + 2));
  }
 
  if(isMulti){

  }else{
    this->cu_h2 = d_MallocHost(tmp * (2*this->k));
    this->cu_h3 = d_MallocHost(tmp * (2*this->k+1));
    this->cu_h4 = d_MallocHost(tmp * (2*this->k+2));
  }

  if(isMulti){

  }else{
    this->cu_d9 = d_Malloc(tmp * (2*this->k + 1));
    this->cu_h5 = d_MallocHost(tmp * (2*this->k+1));
  }
}

cuda::cuda(times *t, unsigned long int size, double restart, unsigned long int size1, unsigned long int size2) : cuda::cuda(t, size, size1, size2){
  if(isMulti){
    std::cout << "XXXXXXXXXXXXXXXX" << std::endl;
  }
  int r = static_cast<int>(restart);
  this->restart = r;
  int tmp = ceil((double)this->size/(double)128);
  this->cu_d10 = d_Malloc(r * size);
  this->cu_d11 = d_Malloc(tmp);
  this->cu_h6 = d_MallocHost(r * size);
  this->cu_h6 = d_MallocHost(tmp);
}

cuda::cuda(times *t, unsigned long int size, int k, double restart, unsigned long int size1, unsigned long int size2) : cuda::cuda(t, size, k, size1, size2){
  if(isMulti){
    std::cout << "XXXXXXXXXXXXXXXX" << std::endl;
  }
  int r = static_cast<int>(restart);
  this->restart = r;
  int tmp = ceil((double)this->size/(double)128);
  this->cu_d10 = d_Malloc(r * size);
  this->cu_d11 = d_Malloc(tmp);
  this->cu_h6 = d_MallocHost(r * size);
  this->cu_h6 = d_MallocHost(tmp);
}

cuda::~cuda(){

  Free(cu_d1);
  Free(cu_d2);
  Free(cu_d3);
  Free(cu_d4);
  Free(cu_d5);
  Free(cu_d6);
  Free(cu_d7);
  Free(cu_d8);
  Free(cu_d9);
  Free(cu_d10);
  Free(cu_d11);

  FreeHost(cu_h1);
  FreeHost(cu_h2);
  FreeHost(cu_h3);
  FreeHost(cu_h4);
  FreeHost(cu_h5);
  FreeHost(cu_h6);
  FreeHost(cu_h7);

  /* Free(cu_d100); */
  /* Free(cu_d101); */
  /* Free(cu_d200); */
  /* Free(cu_d201); */
  FreeHost(cu_h8);

  Free(cu_d1_1);
  Free(cu_d1_2);
  Free(cu_d2_1);
  Free(cu_d2_2);

  Free(cu_d4_1);
  Free(cu_d4_2);
  Free(cu_d5_1);
  Free(cu_d5_2);
}

void cuda::Free(void* ptr){
  checkCudaErrors(hipFree(ptr));
}

void cuda::FreeHost(void* ptr){
  checkCudaErrors(hipHostFree(ptr));
}

void cuda::H2D(double *from, double *to, unsigned long int size, bool timer, int DeviceNum){
  if(timer){
    this->time->start_e();
  }
  checkCudaErrors( hipSetDevice(DeviceNum) );
  checkCudaErrors(hipMemcpy(to, from, sizeof(double)*size, hipMemcpyHostToDevice));
  checkCudaErrors( hipSetDevice(0) );
  if(timer){
    this->time->end_e();
    this->time->h2d_time += this->time->getTime_e();
  }
}

void cuda::D2H(double *from, double *to, unsigned long int size, bool timer, int DeviceNum){
  if(timer){
    this->time->start_e();
  }
  checkCudaErrors( hipSetDevice(DeviceNum) );
  checkCudaErrors(hipMemcpy(to, from, sizeof(double)*size, hipMemcpyDeviceToHost));
  checkCudaErrors( hipSetDevice(0) );
  if(timer){
    this->time->end_e();
    this->time->d2h_time += this->time->getTime_e();
  }
}

void cuda::H2D(int *from, int *to, unsigned long int size, bool timer, int DeviceNum){
  if(timer){
    this->time->start_e();
  }
  checkCudaErrors( hipSetDevice(DeviceNum) );
  checkCudaErrors(hipMemcpy(to, from, sizeof(int)*size, hipMemcpyHostToDevice));
  checkCudaErrors( hipSetDevice(0) );
  if(timer){
    this->time->end_e();
    this->time->h2d_time += this->time->getTime_e();
  }
}

void cuda::D2H(int *from, int *to, unsigned long int size, bool timer, int DeviceNum){
  if(timer){
    this->time->start_e();
  }
  checkCudaErrors( hipSetDevice(DeviceNum) );
  checkCudaErrors(hipMemcpy(to, from, sizeof(int)*size, hipMemcpyDeviceToHost));
  checkCudaErrors( hipSetDevice(0) );
  if(timer){
    this->time->end_e();
    this->time->d2h_time += this->time->getTime_e();
  }
}

/* double* cuda::d_Malloc(unsigned long int size){ */
/*   double *ptr = NULL; */
/*   unsigned long int s = sizeof(double) * size; */
/*   checkCudaErrors(hipMalloc((void**)&ptr, s)); */
/*   return ptr; */
/* } */

double* cuda::d_Malloc(unsigned long int size, int DeviceNum){
  hipSetDevice(DeviceNum);
  double *ptr = NULL;
  unsigned long int s = sizeof(double) * size;
  checkCudaErrors(hipMalloc((void**)&ptr, s));
  hipSetDevice(0);
  return ptr;
}

/* double* cuda::d_MallocHost(unsigned long int size){ */
/*   double *ptr = NULL; */
/*   unsigned long int  s = sizeof(double) * size; */
/*   checkCudaErrors(hipHostMalloc((void**)&ptr, s)); */
/*   return ptr; */
/* } */

double* cuda::d_MallocHost(unsigned long int size, int DeviceNum){
  hipSetDevice(DeviceNum);
  double *ptr = NULL;
  unsigned long int  s = sizeof(double) * size;
  checkCudaErrors(hipHostMalloc((void**)&ptr, s));
  hipSetDevice(0);
  return ptr;
}

/* int* cuda::i_Malloc(unsigned long int size){ */
/*   int *ptr = NULL; */
/*   unsigned long int s = sizeof(int) * size; */
/*   checkCudaErrors(hipMalloc((void**)&ptr, s)); */
/*   return ptr; */
/* } */

int* cuda::i_Malloc(unsigned long int size, int DeviceNum){
  hipSetDevice(DeviceNum);
  int *ptr = NULL;
  unsigned long int s = sizeof(int) * size;
  checkCudaErrors(hipMalloc((void**)&ptr, s));
  hipSetDevice(0);
  return ptr;
}

/* int* cuda::i_MallocHost(unsigned long int size){ */
/*   int *ptr = NULL; */
/*   unsigned long int s = sizeof(int) * size; */
/*   checkCudaErrors(hipHostMalloc((void**)&ptr, s)); */
/*   return ptr; */
/* } */

int* cuda::i_MallocHost(unsigned long int size, int DeviceNum){
  hipSetDevice(DeviceNum);
  int *ptr = NULL;
  unsigned long int s = sizeof(int) * size;
  checkCudaErrors(hipHostMalloc((void**)&ptr, s));
  hipSetDevice(0);
  return ptr;
}

/* void cuda::Memset(double *ptr, double val, unsigned long int size){ */
/*   checkCudaErrors(hipMemset(ptr, val, sizeof(double)*size)); */
/* } */
/*  */
/* void cuda::Memset(int *ptr, int val, unsigned long int size){ */
/*   checkCudaErrors(hipMemset(ptr, val, sizeof(int)*size)); */
/* } */

void cuda::Memset(double *ptr, double val, unsigned long int size, bool timer, int DeviceNum){
  if(timer){
    this->time->start_e();
  }
  hipSetDevice(DeviceNum);
  checkCudaErrors(hipMemset(ptr, val, sizeof(double)*size));
  hipSetDevice(0);
  if(timer){
    this->time->end_e();
    this->time->memset_time += this->time->getTime_e();
  }
}

void cuda::Memset(int *ptr, int val, unsigned long int size, bool timer, int DeviceNum){
  if(timer){
    this->time->start_e();
  }
  hipSetDevice(DeviceNum);
  checkCudaErrors(hipMemset(ptr, val, sizeof(int)*size));
  hipSetDevice(0);
  if(timer){
    this->time->end_e();
    this->time->memset_time += this->time->getTime_e();
  }
}

void cuda::Reset(int DeviceNum){
  /* checkCudaErrors(hipDeviceSynchronize()); */
  /* checkCudaErrors(hipProfilerStop()); */
  std::cout << "Reset Device" << std::endl;
  checkCudaErrors (hipSetDevice(DeviceNum) );
  checkCudaErrors (hipDeviceReset() );
  checkCudaErrors (hipSetDevice(0) );
}

void cuda::MtxVec_mult(double *in, double *out, unsigned long size, double *val, int *col, int *ptr){
  double *D_in = NULL, *D_out = NULL;

  this->time->start();

  std::cout << "Useing defalut MtxVec" << std::endl;

  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  D_in = d_Malloc(size);
  D_out = d_Malloc(size);

  Memset(D_out, 0, size, true);

  H2D(in, D_in, size, true);

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }
  
  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(size, val, col, ptr, D_in, D_out);
  checkCudaErrors( hipPeekAtLastError() );

  D2H(D_out, out, size, true);

  Free(D_in);
  Free(D_out);

  this->time->end();
  this->time->mv_time += this->time->getTime();
}

void cuda::MtxVec_mult(double *in, double *out, double *val, int *col, int *ptr){

  this->time->start();

  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;
  
  Memset(this->cu_d2, 0, size, true);

  //d1 -> in
  //d2 -> out
  H2D(in, this->cu_d1, size, true);


  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d2);
  checkCudaErrors( hipPeekAtLastError() );

  D2H(cu_d2, out, size, true);

  this->time->end();
  this->time->mv_time += this->time->getTime();
}

void cuda::MtxVec_mult(double *in, unsigned long int inindex, unsigned long int insize, double *out, unsigned long int outindex, unsigned long int outsize, double *val, int *col, int *ptr){
  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  this->time->start();

  Memset(this->cu_d2, 0, size, true);

  //d1 -> in
  //d2 -> out
  H2D((double*)(in+(inindex*insize)), this->cu_d1, size, true);

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d2);
  checkCudaErrors( hipPeekAtLastError() );

  D2H(this->cu_d2, (double*)(out+(outindex*outsize)), size, true);

  this->time->end();
  this->time->mv_time += this->time->getTime();
}

void cuda::MtxVec_mult(double *in, unsigned long int inindex, unsigned long int insize, double *out, double *val, int *col, int *ptr){
  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  this->time->start();

  Memset(this->cu_d2, 0, size, true);

  //d1 -> in
  //d2 -> out
  H2D((double*)(in+(inindex*insize)), this->cu_d1, size, true);

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }
  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d2);
  checkCudaErrors( hipPeekAtLastError() );

  D2H(this->cu_d2, out, size, true);

  this->time->end();
  this->time->mv_time += this->time->getTime();

}

double cuda::dot(double *in1, double *in2, unsigned long int size){
  double *D_in1=NULL, *D_in2=NULL;
  double *H_out=NULL, *D_out=NULL, sum=0.0;

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();
  std::cout << "Useing defalut MtxVec" << std::endl;

  D_in1 = d_Malloc(size);
  D_in2 = d_Malloc(size);
  D_out = d_Malloc(BlockPerGrid);
  H_out = new double [BlockPerGrid];


  H2D(in1, D_in1, size);
  H2D(in2, D_in2, size);


  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(size, D_in1, D_in2, D_out);
  checkCudaErrors( hipPeekAtLastError() );

  D2H(D_out, H_out, BlockPerGrid);

/* #pragma omp parallel for schedule(static) reduction(+:sum) */
  for(int i=0; i<BlockPerGrid; i++){
    sum += H_out[i];
  }

  delete[] H_out;
  Free(D_in1);
  Free(D_in2);
  Free(D_out);

  this->time->end();
  this->time->dot_time += this->time->getTime();

  return sum;
}

double cuda::dot(double *in1, double *in2){
  double sum=0.0;

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();

  Memset(this->cu_d3, 0, BlockPerGrid, true);

  //d_1 -> in1
  //d_2 -> in2
  //d_3 -> out
  H2D(in1, this->cu_d1, size, true);
  H2D(in2, this->cu_d2, size, true);


  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d1, cu_d2, cu_d3);
  checkCudaErrors( hipPeekAtLastError() );


  //d_3 -> out
  //h_1 -> out(host)
  D2H(cu_d3, cu_h1, BlockPerGrid, true);

/* #pragma omp parallel for schedule(static) reduction(+:sum) */
  for(int i=0; i<BlockPerGrid; i++){
    sum += cu_h1[i];
  }

  this->time->end();
  this->time->dot_time += this->time->getTime();

  return sum;
}


double cuda::dot(double *in1, unsigned long int in1index, unsigned long int in1size, double *in2, unsigned long int in2index, unsigned long int in2size){
  double sum=0.0;

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();

  Memset(this->cu_d3, 0, BlockPerGrid, true);

  //d_1 -> in1
  //d_2 -> in2
  //d_3 -> out
  H2D((double*)(in1+(in1index*in1size)), this->cu_d1, size, true);
  H2D((double*)(in2+(in2index*in2size)), this->cu_d2, size, true);

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d1, cu_d2, cu_d3);
  checkCudaErrors( hipPeekAtLastError() );


  //d_3 -> out
  //h_1 -> out(host)
  D2H(cu_d3, cu_h1, BlockPerGrid, true);

/* #pragma omp parallel for schedule(static) reduction(+:sum) */
  for(int i=0; i<BlockPerGrid; i++){
    sum += cu_h1[i];
  }

  this->time->end();
  this->time->dot_time += this->time->getTime();

  return sum;

}

double cuda::dot(double *in1, double *in2, unsigned long int in2index, unsigned long int in2size){
  double sum=0.0;

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();

  Memset(this->cu_d3, 0, BlockPerGrid, true);

  //d_1 -> in1
  //d_2 -> in2
  //d_3 -> out
  H2D(in1, this->cu_d1, size, true);
  H2D((double*)(in2+(in2index*in2size)), this->cu_d2, size, true);


  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d1, cu_d2, cu_d3);
  checkCudaErrors( hipPeekAtLastError() );


  //d_3 -> out
  //h_1 -> out(host)
  D2H(cu_d3, cu_h1, BlockPerGrid, true);

/* #pragma omp parallel for schedule(static) reduction(+:sum) */
  for(int i=0; i<BlockPerGrid; i++){
    sum += cu_h1[i];
  }

  this->time->end();
  this->time->dot_time += this->time->getTime();
  return sum;
}

double cuda::dot(double *in1, unsigned long int in1index, unsigned long int in1size, double *in2){
  double sum=0.0;

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();

  Memset(this->cu_d3, 0, BlockPerGrid, true);

  //d_1 -> in1
  //d_2 -> in2
  //d_3 -> out
  H2D((double*)(in1+(in1index*in1size)), this->cu_d2, size, true);
  H2D(in2, this->cu_d2, size, true);


  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d1, cu_d2, cu_d3);
  checkCudaErrors( hipPeekAtLastError() );


  //d_3 -> out
  //h_1 -> out(host)
  D2H(cu_d3, cu_h1, BlockPerGrid, true);


/* #pragma omp parallel for schedule(static) reduction(+:sum) */
  for(int i=0; i<BlockPerGrid; i++){
    sum += cu_h1[i];
  }

  this->time->end();
  this->time->dot_time += this->time->getTime();

  return sum;
}

void cuda::CSR2CSC(double *dCSRval, int *dCSRcol, int *dCSRptr, double *CSCval, int *CSCrow, int *CSCptr, double *dCSCval, int *dCSCrow, int *dCSCptr, unsigned long int N, unsigned long int NNZ){
  hipsparseHandle_t handle=0;
  hipsparseCreate(&handle);

  hipsparseStatus_t status = hipsparseDcsr2csc(handle, N, N, NNZ, dCSRval, dCSRptr, dCSRcol, dCSCval, dCSCrow, dCSCptr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
  std::cout << GREEN << "[○] Done" << RESET << std::endl;

  if(status != HIPSPARSE_STATUS_SUCCESS){
    std::cout << "error in cusparse CSR2CSC cuda" << std::endl;
    if(status == HIPSPARSE_STATUS_NOT_INITIALIZED){
      std::cout << "HIPSPARSE_STATUS_NOT_INITIALIZED" << std::endl;
    }else if(status == HIPSPARSE_STATUS_ALLOC_FAILED){
      std::cout << "HIPSPARSE_STATUS_ALLOC_FAILED" << std::endl;
    }else if(status == HIPSPARSE_STATUS_INVALID_VALUE){
      std::cout << "HIPSPARSE_STATUS_INVALID_VALUE" << std::endl;
    }else if(status == HIPSPARSE_STATUS_ARCH_MISMATCH){
      std::cout << "HIPSPARSE_STATUS_ARCH_MISMATCH" << std::endl;
    }else if(status == HIPSPARSE_STATUS_MAPPING_ERROR){
      std::cout << "HIPSPARSE_STATUS_MAPPING_ERROR" << std::endl;
    }else if(status == HIPSPARSE_STATUS_EXECUTION_FAILED){
      std::cout << "HIPSPARSE_STATUS_EXECUTION_FAILED" << std::endl;
    }else if(status == HIPSPARSE_STATUS_INTERNAL_ERROR){
      std::cout << "HIPSPARSE_STATUS_INTERNAL_ERROR" << std::endl;
    }else if(status == HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED){
      std::cout << "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED" << std::endl;
    }else{
      std::cout << "else error" << std::endl;
    }
    exit(-1);
  }

  hipMemcpy(CSCval, dCSCval, sizeof(double)*NNZ, hipMemcpyDeviceToHost);
  hipMemcpy(CSCrow, dCSCrow, sizeof(int)*NNZ, hipMemcpyDeviceToHost);
  hipMemcpy(CSCptr, dCSCptr, sizeof(int)*(N+1), hipMemcpyDeviceToHost);

}

void cuda::CSR2CSC(double *CSRval, int *CSRcol, int *CSRptr, double *CSCval, int *CSCrow, int *CSCptr, unsigned long int N, unsigned long int NNZ){

  double *dCSRval;
  int *dCSRcol, *dCSRptr;
  double *dCSCval;
  int *dCSCrow, *dCSCptr;
  hipsparseHandle_t handle=0;
  hipsparseCreate(&handle);


  hipMalloc((void**)&dCSRval, sizeof(double)*NNZ);
  hipMalloc((void**)&dCSRcol, sizeof(int)*NNZ);
  hipMalloc((void**)&dCSRptr, sizeof(int)*(N+1));

  hipMalloc((void**)&dCSCval, sizeof(double)*NNZ);
  hipMalloc((void**)&dCSCrow, sizeof(int)*NNZ);
  hipMalloc((void**)&dCSCptr, sizeof(int)*(N+1));

  hipMemcpy(dCSRval, CSRval, sizeof(double)*NNZ, hipMemcpyHostToDevice);
  hipMemcpy(dCSRcol, CSRcol, sizeof(int)*NNZ, hipMemcpyHostToDevice );
  hipMemcpy(dCSRptr, CSRptr, sizeof(int)*(N+1),  hipMemcpyHostToDevice);

  memset(CSCval, 0, sizeof(double)*NNZ);
  memset(CSCrow, 0, sizeof(int)*NNZ);
  memset(CSCptr, 0, sizeof(int)*(N+1));

  hipMemset(dCSCval, 0, sizeof(double)*NNZ);
  hipMemset(dCSCrow, 0, sizeof(int)*NNZ);
  hipMemset(dCSCptr, 0, sizeof(int)*(N+1));

  hipsparseStatus_t status = hipsparseDcsr2csc(handle, N, N, NNZ, dCSRval, dCSRptr, dCSRcol, dCSCval, dCSCrow, dCSCptr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
  std::cout << GREEN << "[○] Done" << RESET << std::endl;

  if(status != HIPSPARSE_STATUS_SUCCESS){
    std::cout << "error in cusparse CSR2CSC CPU" << std::endl;
    exit(-1);
  }

  hipMemcpy(CSCval, dCSCval, sizeof(double)*NNZ, hipMemcpyDeviceToHost);
  hipMemcpy(CSCrow, dCSCrow, sizeof(int)*NNZ, hipMemcpyDeviceToHost);
  hipMemcpy(CSCptr, dCSCptr, sizeof(int)*(N+1), hipMemcpyDeviceToHost);

  hipFree(dCSRval);
  hipFree(dCSRcol);
  hipFree(dCSRptr);

  hipFree(dCSCval);
  hipFree(dCSCrow);
  hipFree(dCSCptr);
}

void cuda::Kskip_cg_bicg_base(double *Ar, double *Ap, double *rvec, double *pvec, const int kskip, double *val, int *col, int *ptr){
  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  this->time->start();

  Memset(this->cu_d4, 0, size*(2*k+1), true);
  Memset(this->cu_d5, 0, size*(2*k+2), true);

  //r -> d1
  //p -> d2
  H2D(rvec, this->cu_d1, size, true);
  H2D(pvec, this->cu_d2, size, true);

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  // d1(in) --> d4(out)
  // d2(in) --> d5(out)

  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d4, 0, this->size);
  checkCudaErrors( hipPeekAtLastError() );
  
  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d2, cu_d5, 0, this->size);
  checkCudaErrors( hipPeekAtLastError() );
 
  for(int i=1; i<2*kskip+2; i++){
    if(i<2*kskip+1){
      kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d4, i-1, this->size, cu_d4, i, this->size);
      checkCudaErrors( hipPeekAtLastError() );
    }
    kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d5, i-1, this->size, cu_d5, i, this->size);
    checkCudaErrors( hipPeekAtLastError() );
  }


  D2H(this->cu_d4, Ar, size*(2*kskip+1), true);
  D2H(this->cu_d5, Ap, size*(2*kskip+2), true);

  this->time->end();
  this->time->mv_time += this->time->getTime();
}

void cuda::Kskip_cg_bicg_base2(double *Ar, double *Ap, double *rvec, double *pvec, const int kskip, double *val, int *col, int *ptr){
  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  this->time->start();

  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipMemsetAsync(cu_d4, 0, size*(2*kskip+1), stream1);

  hipMemcpyAsync(this->cu_d1, rvec, sizeof(double)*size, hipMemcpyHostToDevice, stream1);

  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock, 0, stream1>>>(this->size, val, col, ptr, cu_d1, cu_d4, 0, this->size);

  hipMemcpyAsync((double*)(Ar+(0*this->size)), (double*)(cu_d4+(0*this->size)), sizeof(double)*size, hipMemcpyDeviceToHost, stream1);

  hipMemsetAsync(cu_d5, 0, size*(2*kskip+2), stream2);

  hipMemcpyAsync(this->cu_d2, pvec, sizeof(double)*size, hipMemcpyHostToDevice, stream2);

  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock, 0, stream2>>>(this->size, val, col, ptr, cu_d2, cu_d5, 0, this->size);

  hipMemcpyAsync((double*)(Ap+(0*this->size)), (double*)(cu_d5+(0*this->size)), sizeof(double)*size, hipMemcpyDeviceToHost, stream2);

  
  for(int i=1; i<2*kskip+2; i++){
    if(i<2*kskip+1){
      kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock, 0, stream1>>>(this->size, val, col, ptr, cu_d4, i-1, this->size, cu_d4, i, this->size);

      hipMemcpyAsync((double*)(Ar+(i*this->size)), (double*)(cu_d4+(i*this->size)), sizeof(double)*size, hipMemcpyDeviceToHost, stream1);
    }

    kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock, 0, stream2>>>(this->size, val, col, ptr, cu_d5, i-1, this->size, cu_d5, i, this->size);

    hipMemcpyAsync((double*)(Ap+(i*this->size)), (double*)(cu_d5+(i*this->size)), sizeof(double)*size, hipMemcpyDeviceToHost, stream2);
  }

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);

  this->time->end();
  this->time->mv_time += this->time->getTime();

}

void cuda::Kskip_cg_innerProduce(double *delta, double *eta, double *zeta, double *Ar, double *Ap, double *rvec, double *pvec, int kskip, double *val, int *col, int *ptr){

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();

  Memset(this->cu_d6, 0, BlockPerGrid * (2*kskip));
  Memset(this->cu_d7, 0, BlockPerGrid * (2*kskip+1));
  Memset(this->cu_d8, 0, BlockPerGrid * (2*kskip+2));

  //d1 -> r
  //d2 -> p
  //d4 -> Ar
  //d5 -> Ap

  //d6 -> delta
  //d7 -> eta
  //d8 -> zeta

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  for(int i=0; i<2*kskip+2; i++){
    if(i<2*kskip){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d4, i, this->size, cu_d1, cu_d6, i, BlockPerGrid);
    }
    if(i<2*kskip+1){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d5, i, this->size, cu_d1, cu_d7, i, BlockPerGrid);
    }
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d5, i, this->size, cu_d2, cu_d8, i, BlockPerGrid);
  }


  //d6 -> delta -> h2
  //d7 -> eta -> h3
  //d8 -> zeta -> h4

  D2H(cu_d6, cu_h2, BlockPerGrid * (2*kskip));
  D2H(cu_d7, cu_h3, BlockPerGrid * (2*kskip+1));
  D2H(cu_d8, cu_h4, BlockPerGrid * (2*kskip+2));

  double tmp1 = 0.0;
  double tmp2 = 0.0;
  double tmp3 = 0.0;
/* #pragma omp parallel for reduction(+:tmp1, tmp2, tmp3) schedule(static) firstprivate(delta, eta, zeta, cu_h2, cu_h3, cu_h4) lastprivate(delta, eta, zeta) */
  for(int i=0; i<2*kskip+2; i++){
    tmp1 = 0.0;
    tmp2 = 0.0;
    tmp3 = 0.0;
    if(i<2*kskip){
      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h2[i*BlockPerGrid+j];
      }
      delta[i] = tmp1;
    }
    if(i<2*kskip+1){
      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h3[i*BlockPerGrid+j];
      }
      eta[i] = tmp2;
    }
    for(int j=0; j<BlockPerGrid; j++){
      tmp3 += cu_h4[i*BlockPerGrid+j];
    }
    zeta[i] = tmp3;
  }
  this->time->end();
  this->time->dot_time += this->time->getTime();
}

void cuda::Kskip_cg_innerProduce2(double *delta, double *eta, double *zeta, double *Ar, double *Ap, double *rvec, double *pvec, int kskip, double *val, int *col, int *ptr){

  this->time->start();

  hipStream_t stream1, stream2, stream3;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  hipMemsetAsync(cu_d6, 0, sizeof(double)*BlockPerGrid*(2*kskip), stream1);
  hipMemsetAsync(cu_d7, 0, sizeof(double)*BlockPerGrid*(2*kskip+1), stream2);
  hipMemsetAsync(cu_d8, 0, sizeof(double)*BlockPerGrid*(2*kskip+2), stream3);

  //d1 -> r
  //d2 -> p
  //d4 -> Ar
  //d5 -> Ap

  //d6 -> delta
  //d7 -> eta
  //d8 -> zeta

double tmp1 = 0.0;
double tmp2 = 0.0;
double tmp3 = 0.0;

  for(int i=0; i<2*kskip+2; i++){
    tmp1 = 0.0;
    tmp2 = 0.0;
    tmp3 = 0.0;
    if(i<2*kskip){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream1>>>(this->size, cu_d4, i, this->size, cu_d1, cu_d6, i, BlockPerGrid);

      hipMemcpyAsync((double*)(cu_h2+(i*BlockPerGrid)), (double*)(cu_d6+(i*BlockPerGrid)), sizeof(double)*(BlockPerGrid), hipMemcpyDeviceToHost, stream1);
      hipStreamSynchronize(stream1);

      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h2[i*BlockPerGrid+j];
      }
      delta[i] = tmp1;

    }
    if(i<2*kskip+1){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream2>>>(this->size, cu_d5, i, this->size, cu_d1, cu_d7, i, BlockPerGrid);

      hipMemcpyAsync((double*)(cu_h3+(i*BlockPerGrid)), (double*)(cu_d7+(i*BlockPerGrid)), sizeof(double)*(BlockPerGrid), hipMemcpyDeviceToHost, stream2);
      hipStreamSynchronize(stream2);

      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h3[i*BlockPerGrid+j];
      }
      eta[i] = tmp2;

    }
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream3>>>(this->size, cu_d5, i, this->size, cu_d2, cu_d8, i, BlockPerGrid);

    hipMemcpyAsync((double*)(cu_h4+(i*BlockPerGrid)), (double*)(cu_d8+(i*BlockPerGrid)), sizeof(double)*(BlockPerGrid), hipMemcpyDeviceToHost, stream3);

    hipStreamSynchronize(stream3);

    for(int j=0; j<BlockPerGrid; j++){
      tmp3 += cu_h4[i*BlockPerGrid+j];
    }
    zeta[i] = tmp3;

  }
  this->time->end();
  this->time->dot_time += this->time->getTime();
}

void cuda::Kskip_cg_innerProduce3(double *delta, double *eta, double *zeta, double *Ar, double *Ap, double *rvec, double *pvec, int kskip, double *val, int *col, int *ptr){
  hipStream_t stream1, stream2, stream3;

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();

  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);


  hipMemsetAsync(cu_d6, 0, sizeof(double)*BlockPerGrid*(2*kskip), stream1);
  hipMemsetAsync(cu_d7, 0, sizeof(double)*BlockPerGrid*(2*kskip+1), stream2);
  hipMemsetAsync(cu_d8, 0, sizeof(double)*BlockPerGrid*(2*kskip+2), stream3);


  //d1 -> r
  //d2 -> p
  //d4 -> Ar
  //d5 -> Ap

  //d6 -> delta
  //d7 -> eta
  //d8 -> zeta

  hipDeviceSynchronize();
  
  double tmp1 = 0.0;
  double tmp2 = 0.0;
  double tmp3 = 0.0;

  for(int i=0; i<2*kskip+2; i++){
    tmp1 = 0.0;
    tmp2 = 0.0;
    tmp3 = 0.0;
    if(i<2*kskip){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream1>>>(this->size, cu_d4, i, this->size, cu_d1, cu_d6, i, BlockPerGrid);

      if(i==2*kskip-1){
        hipMemcpyAsync(cu_h2, cu_d6, sizeof(double)*(BlockPerGrid)*(2*kskip), hipMemcpyDeviceToHost, stream1);
      }

    }
    if(i<2*kskip+1){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream2>>>(this->size, cu_d5, i, this->size, cu_d1, cu_d7, i, BlockPerGrid);

      if(i==2*kskip){
        hipMemcpyAsync(cu_h3, cu_d7, sizeof(double)*(BlockPerGrid)*(2*kskip+1), hipMemcpyDeviceToHost, stream2);
      }
    }
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream3>>>(this->size, cu_d5, i, this->size, cu_d2, cu_d8, i, BlockPerGrid);

    if(i==2*kskip+1){
      hipMemcpyAsync(cu_h4, cu_d8, sizeof(double)*(BlockPerGrid)*(2*kskip+2), hipMemcpyDeviceToHost, stream3);
    }

    hipDeviceSynchronize();
  }

/* #pragma omp parallel for reduction(+:tmp1, tmp2, tmp3) schedule(static) firstprivate(delta, eta, zeta, cu_h2, cu_h3, cu_h4) lastprivate(delta, eta, zeta) */
  for(int i=0; i<2*kskip+2; i++){
    tmp1 = 0.0;
    tmp2 = 0.0;
    tmp3 = 0.0;
    if(i<2*kskip){
      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h2[i*BlockPerGrid+j];
      }
      delta[i] = tmp1;
    }
    if(i<2*kskip+1){
      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h3[i*BlockPerGrid+j];
      }
      eta[i] = tmp2;
    }
    for(int j=0; j<BlockPerGrid; j++){
      tmp3 += cu_h4[i*BlockPerGrid+j];
    }
    zeta[i] = tmp3;
  }

  this->time->end();
  this->time->dot_time += this->time->getTime();
}

void cuda::Kskip_bicg_innerProduce(double *theta, double *eta, double *rho, double *phi, double *Ar, double *Ap, double *r_vec, double *p_vec, int kskip, double *val, int *col, int *ptr){

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();

  Memset(this->cu_d6, 0, BlockPerGrid * (2*kskip), true);
  Memset(this->cu_d7, 0, BlockPerGrid * (2*kskip+1), true);
  Memset(this->cu_d9, 0, BlockPerGrid * (2*kskip+1), true);
  Memset(this->cu_d8, 0, BlockPerGrid * (2*kskip+2), true);


  H2D(r_vec, cu_d1, this->size, true);
  H2D(p_vec, cu_d2, this->size, true);

  //d1 -> *r
  //d2 -> *p
  //d4 -> Ar
  //d5 -> Ap

  //d6 -> theta
  //d7 -> eta
  //d9 -> rho
  //d8 -> phi

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  for(int i=0; i<2*kskip+2; i++){
    if(i<2*kskip){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d4, i, this->size, cu_d1, cu_d6, i, BlockPerGrid);
    }
    if(i<2*kskip+1){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d5, i, this->size, cu_d1, cu_d7, i, BlockPerGrid);
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d4, i, this->size, cu_d2, cu_d9, i, BlockPerGrid);
    }
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d5, i, this->size, cu_d2, cu_d8, i, BlockPerGrid);
  }


  //d6 -> theta -> h2
  //d7 -> eta -> h3
  //d9 -> rho -> h5
  //d8 -> phi -> h4

  D2H(cu_d6, cu_h2, BlockPerGrid * (2*kskip), true);
  D2H(cu_d7, cu_h3, BlockPerGrid * (2*kskip+1), true);
  D2H(cu_d9, cu_h5, BlockPerGrid * (2*kskip+1), true);
  D2H(cu_d8, cu_h4, BlockPerGrid * (2*kskip+2), true);


  double tmp1 = 0.0;
  double tmp2 = 0.0;
  double tmp3 = 0.0;
  double tmp4 = 0.0;
/* #pragma omp parallel for reduction(+:tmp1, tmp2, tmp3, tmp4) schedule(static) firstprivate(theta, eta, rho, phi, cu_h2, cu_h3, cu_h4, cu_h5) lastprivate(theta, eta, rho, phi) */
  for(int i=0; i<2*kskip+2; i++){
    tmp1 = 0.0;
    tmp2 = 0.0;
    tmp3 = 0.0;
    tmp4 = 0.0;
    if(i<2*kskip){
      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h2[i*BlockPerGrid+j];
      }
      theta[i] = tmp1;
    }
    if(i<2*kskip+1){
      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h3[i*BlockPerGrid+j];
        tmp3 += cu_h5[i*BlockPerGrid+j];
      }
      eta[i] = tmp2;
      rho[i] = tmp3;
    }
    for(int j=0; j<BlockPerGrid; j++){
      tmp4 += cu_h4[i*BlockPerGrid+j];
    }
    phi[i] = tmp4;
  }

  this->time->end();
  this->time->dot_time += this->time->getTime();

}

void cuda::Kskip_bicg_innerProduce2(double *theta, double *eta, double *rho, double *phi, double *Ar, double *Ap, double *r_vec, double *p_vec, int kskip, double *val, int *col, int *ptr){
  //d1 -> *r
  //d2 -> *p
  //d4 -> Ar
  //d5 -> Ap

  //d6 -> theta
  //d7 -> eta
  //d9 -> rho
  //d8 -> phi

  //d6 -> theta -> h2
  //d7 -> eta -> h3
  //d9 -> rho -> h5
  //d8 -> phi -> h4

  hipStream_t stream1, stream2, stream3, stream4;

  this->time->start();

  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);
  hipStreamCreate(&stream4);

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  hipMemsetAsync(cu_d6, 0, sizeof(double)*BlockPerGrid*(2*kskip), stream1);
  hipMemsetAsync(cu_d7, 0, sizeof(double)*BlockPerGrid*(2*kskip+1), stream2);
  hipMemsetAsync(cu_d9, 0, sizeof(double)*BlockPerGrid*(2*kskip+1), stream3);


  hipMemcpyAsync(cu_d1, r_vec, sizeof(double)*size, hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(cu_d2, p_vec, sizeof(double)*size, hipMemcpyHostToDevice, stream2);

  double tmp1 = 0.0;
  double tmp2 = 0.0;
  double tmp3 = 0.0;
  double tmp4 = 0.0;
  for(int i=0; i<2*kskip+2; i++){
    tmp1 = 0.0;
    tmp2 = 0.0;
    tmp3 = 0.0;
    tmp4 = 0.0;
    if(i<2*kskip){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream1>>>(this->size, cu_d4, i, this->size, cu_d1, cu_d6, i, BlockPerGrid);
      hipMemcpyAsync((double*)(cu_h2+(i*BlockPerGrid)), (double*)(cu_d6+(i*BlockPerGrid)), sizeof(double)*(BlockPerGrid), hipMemcpyDeviceToHost, stream1);
      hipStreamSynchronize(stream1);
/* #pragma omp parallel for reduction(+:tmp1) schedule(static) */
      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h2[i*BlockPerGrid+j];
      }
      theta[i] = tmp1;
    }
    if(i<2*kskip+1){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream2>>>(this->size, cu_d5, i, this->size, cu_d1, cu_d7, i, BlockPerGrid);
      hipMemcpyAsync((double*)(cu_h3+(i*BlockPerGrid)), (double*)(cu_d7+(i*BlockPerGrid)), sizeof(double)*(BlockPerGrid), hipMemcpyDeviceToHost, stream2);
      hipStreamSynchronize(stream2);
/* #pragma omp parallel for reduction(+:tmp2) schedule(static) */
      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h3[i*BlockPerGrid+j];
      }
      eta[i] = tmp2;
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream3>>>(this->size, cu_d4, i, this->size, cu_d2, cu_d9, i, BlockPerGrid);
      hipMemcpyAsync((double*)(cu_h5+(i*BlockPerGrid)), (double*)(cu_d9+(i*BlockPerGrid)), sizeof(double)*(BlockPerGrid), hipMemcpyDeviceToHost, stream3);
      hipStreamSynchronize(stream3);
/* #pragma omp parallel for reduction(+:tmp3) schedule(static) */
      for(int j=0; j<BlockPerGrid; j++){
        tmp3 += cu_h5[i*BlockPerGrid+j];
      }
      rho[i] = tmp3;
    }
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream4>>>(this->size, cu_d5, i, this->size, cu_d2, cu_d8, i, BlockPerGrid);
    hipMemcpyAsync((double*)(cu_h4+(i*BlockPerGrid)), (double*)(cu_d8+(i*BlockPerGrid)), sizeof(double)*(BlockPerGrid), hipMemcpyDeviceToHost, stream4);
    hipStreamSynchronize(stream4);
/* #pragma omp parallel for reduction(+:tmp4) schedule(static) */
    for(int j=0; j<BlockPerGrid; j++){
      tmp4 += cu_h4[i*BlockPerGrid+j];
    }
    phi[i] = tmp4;
  }

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);
  hipStreamDestroy(stream4);

  this->time->end();
  this->time->dot_time += this->time->getTime();
}

void cuda::Kskip_bicg_innerProduce3(double *theta, double *eta, double *rho, double *phi, double *Ar, double *Ap, double *r_vec, double *p_vec, int kskip, double *val, int *col, int *ptr){
  //d1 -> *r
  //d2 -> *p
  //d4 -> Ar
  //d5 -> Ap

  //d6 -> theta
  //d7 -> eta
  //d9 -> rho
  //d8 -> phi

  //d6 -> theta -> h2
  //d7 -> eta -> h3
  //d9 -> rho -> h5
  //d8 -> phi -> h4

  hipStream_t stream1, stream2, stream3, stream4;

  this->time->start();

  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);
  hipStreamCreate(&stream4);

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  hipMemsetAsync(cu_d6, 0, sizeof(double)*BlockPerGrid*(2*kskip), stream1);
  hipMemsetAsync(cu_d7, 0, sizeof(double)*BlockPerGrid*(2*kskip+1), stream2);
  hipMemsetAsync(cu_d9, 0, sizeof(double)*BlockPerGrid*(2*kskip+1), stream3);
  hipMemsetAsync(cu_d8, 0, sizeof(double)*BlockPerGrid*(2*kskip+2), stream4);

  hipMemcpyAsync(cu_d1, r_vec, sizeof(double)*size, hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(cu_d2, p_vec, sizeof(double)*size, hipMemcpyHostToDevice, stream2);

  hipDeviceSynchronize();

  double tmp1 = 0.0;
  double tmp2 = 0.0;
  double tmp3 = 0.0;
  double tmp4 = 0.0;
  for(int i=0; i<2*kskip+2; i++){
    if(i<2*kskip){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream1>>>(this->size, cu_d4, i, this->size, cu_d1, cu_d6, i, BlockPerGrid);
      if(i==2*kskip-1){
        hipMemcpyAsync(cu_h2, cu_d6, sizeof(double)*(BlockPerGrid*(2*kskip)), hipMemcpyDeviceToHost, stream1);
      }
    }
    if(i<2*kskip+1){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream2>>>(this->size, cu_d5, i, this->size, cu_d1, cu_d7, i, BlockPerGrid);
      if(i==2*kskip){
        hipMemcpyAsync(cu_h3, cu_d7, sizeof(double)*(BlockPerGrid*(2*kskip+1)), hipMemcpyDeviceToHost, stream2);
      }
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream3>>>(this->size, cu_d4, i, this->size, cu_d2, cu_d9, i, BlockPerGrid);
      if(i==2*kskip){
        hipMemcpyAsync(cu_h5, cu_d9, sizeof(double)*(BlockPerGrid*(2*kskip+1)), hipMemcpyDeviceToHost, stream3);
      }
    }
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream4>>>(this->size, cu_d5, i, this->size, cu_d2, cu_d8, i, BlockPerGrid);
    if(i==2*kskip+1){
      hipMemcpyAsync(cu_h4, cu_d8, sizeof(double)*(BlockPerGrid*(2*kskip+2)), hipMemcpyDeviceToHost, stream4);
    }
    hipDeviceSynchronize();
  }

/* #pragma omp parallel for reduction(+:tmp1, tmp2, tmp3, tmp4) schedule(static) firstprivate(theta, eta, rho, phi, cu_h2, cu_h3, cu_h4, cu_h5) lastprivate(theta, eta, rho, phi) */
  for(int i=0; i<2*kskip+2; i++){
    tmp1 = 0.0;
    tmp2 = 0.0;
    tmp3 = 0.0;
    tmp4 = 0.0;
    if(i<2*kskip){
      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h2[i*BlockPerGrid+j];
      }
      theta[i] = tmp1;
    }
    if(i<2*kskip+1){
      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h3[i*BlockPerGrid+j];
        tmp3 += cu_h5[i*BlockPerGrid+j];
      }
      eta[i] = tmp2;
      rho[i] = tmp3;
    }
    for(int j=0; j<BlockPerGrid; j++){
      tmp4 += cu_h4[i*BlockPerGrid+j];
    }
    phi[i] = tmp4;
  }
  
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);
  hipStreamDestroy(stream4);

  this->time->end();
  this->time->dot_time += this->time->getTime();
}

void cuda::dot_gmres(double *wvec, double *vmtx, double *hmtx, int k, unsigned long int N){
  this->time->start();
  for(int i=0; i<=k; i++){
    hmtx[k+i*N] = dot(wvec, vmtx, i, N);
  }
  this->time->end();
  this->time->dot_time += this->time->getTime();
}

void cuda::dot_gmres2(double *wvec, double *vmtx, double *hmtx, int k, unsigned long int N){
  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);


  hipStream_t stream1, stream2, stream3;

  this->time->start();

  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);
  double tmp1=0;
  double tmp2=0;

  hipMemcpyAsync(cu_d1, wvec, sizeof(double)*size, hipMemcpyHostToDevice, stream1);
  hipStreamSynchronize(stream1);

  if(k==0){
    //hmtx[0] = dot(wvec, vmtx);
    hipMemsetAsync(cu_d3, 0, BlockPerGrid, stream1);
    hipMemcpyAsync(cu_d2, (double*)(vmtx+(0*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream1);
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream1>>>(this->size, cu_d1, cu_d2, cu_d3);
    hipMemcpyAsync(cu_h1, cu_d3, sizeof(double)*BlockPerGrid, hipMemcpyDeviceToHost, stream1);
    hipStreamSynchronize(stream1);
    for(int i=0; i<BlockPerGrid; i++){
      tmp1 += cu_h1[i];
    }
    hmtx[0] = tmp1;
  }else if((k+1)%2 == 0){
    //even
    /* for(int i=0; i<=k; i++){ */
    /*   hmtx[k+i*size] = dot(wvec, vmtx, i, size); */
    /* } */
    for(int i=0; i<=k; i+=2){
      tmp1 = 0;
      tmp2 = 0;
      hipMemsetAsync(cu_d3, 0, BlockPerGrid, stream1);
      hipMemcpyAsync(cu_d2, (double*)(vmtx+(i*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream1);
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream1>>>(this->size, cu_d1, cu_d2, cu_d3);
      hipMemcpyAsync(cu_h1, cu_d3, sizeof(double)*BlockPerGrid, hipMemcpyDeviceToHost, stream1);
      hipStreamSynchronize(stream1);
      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h1[j];
      }
      hmtx[k+i*size] = tmp1;
      hipMemsetAsync(cu_d101, 0, BlockPerGrid, stream2);
      hipMemcpyAsync(cu_d100, (double*)(vmtx+((i+1)*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream2);
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream2>>>(this->size, cu_d1, cu_d100, cu_d101);
      hipMemcpyAsync(cu_h1, cu_d101, sizeof(double)*BlockPerGrid, hipMemcpyDeviceToHost, stream2);
      hipStreamSynchronize(stream2);
      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h1[j];
      }
      hmtx[k+(i+1)*size] = tmp2;
    }

  }else{
    //odd
    /* for(int i=0; i<=k; i++){ */
    /*   hmtx[k+i*size] = dot(wvec, vmtx, i, size); */
    /* } */
    for(int i=0; i<=k-1; i+=2){
      tmp1 = 0;
      tmp2 = 0;
      hipMemsetAsync(cu_d3, 0, BlockPerGrid, stream1);
      hipMemcpyAsync(cu_d2, (double*)(vmtx+(i*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream1);
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream1>>>(this->size, cu_d1, cu_d2, cu_d3);
      hipMemcpyAsync(cu_h1, cu_d3, sizeof(double)*BlockPerGrid, hipMemcpyDeviceToHost, stream1);
      hipStreamSynchronize(stream1);
      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h1[j];
      }
      hmtx[k+i*size] = tmp1;
      hipMemsetAsync(cu_d101, 0, BlockPerGrid, stream2);
      hipMemcpyAsync(cu_d100, (double*)(vmtx+((i+1)*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream2);
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream2>>>(this->size, cu_d1, cu_d100, cu_d101);
      hipMemcpyAsync(cu_h1, cu_d101, sizeof(double)*BlockPerGrid, hipMemcpyDeviceToHost, stream2);
      hipStreamSynchronize(stream2);
      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h1[j];
      }
      hmtx[k+(i+1)*size] = tmp2;
    }

    tmp1 = 0;
    hipMemsetAsync(cu_d3, 0, BlockPerGrid, stream1);
    hipMemcpyAsync(cu_d2, (double*)(vmtx+(k*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream1);
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream1>>>(this->size, cu_d1, cu_d2, cu_d3);
    hipMemcpyAsync(cu_h1, cu_d3, sizeof(double)*BlockPerGrid, hipMemcpyDeviceToHost, stream1);
    hipStreamSynchronize(stream1);
    for(int i=0; i<BlockPerGrid; i++){
      tmp1 += cu_h1[i];
    }
    hmtx[k+k*size] = tmp1;
  }

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);

  this->time->end();
  this->time->dot_time += this->time->getTime();

}

void cuda::dot_gmres3(double *wvec, double *vmtx, double *hmtx, int k, unsigned long int N){
  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);


  this->time->start();
  
  hipStream_t stream[32];
  for(int i=0; i<32; i++){
    hipStreamCreate(&stream[i]);
  }

  hipMemcpyAsync(cu_d1, wvec, sizeof(double)*size, hipMemcpyHostToDevice, stream[0]);
  hipMemsetAsync(cu_d201, 0, sizeof(double)*BlockPerGrid*(k+1), stream[1]);
  hipStreamSynchronize(stream[0]);
  hipStreamSynchronize(stream[1]);


  double tmp1=0.0;


  if(k==0){
    hmtx[0] = dot(wvec, vmtx);
  }else if((k+1)%2 == 0){
    //even
    /* for(int i=0; i<=k; i++){ */
    /*   hmtx[k+i*size] = dot(wvec, vmtx, i, size); */
    /* } */
    if(k+1 <= 32){
      for(int i=0; i<k+1; i++){
        hipMemcpyAsync((double*)(cu_d200+(i*size)), (double*)(vmtx+(i*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream[i]);
        kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream[i]>>>(this->size, cu_d1, (double*)(cu_d200+(i*size)), (double*)(cu_d201+(i*BlockPerGrid)));
      }
      hipDeviceSynchronize();
      hipMemcpyAsync(cu_h8, cu_d201, sizeof(double)*BlockPerGrid*(k+1), hipMemcpyDeviceToHost, stream[0]);
      hipDeviceSynchronize();

/* #pragma omp parallel for reduction(+:tmp1) schedule(static) firstprivate(hmtx, eta, cu_h8) lastprivate(hmtx) */
      for(int j=0; j<k+1; j++){
        tmp1 = 0.0;
        for(int q=0; q<BlockPerGrid; q++){
          tmp1 += cu_h8[j*BlockPerGrid+q];
        }
        hmtx[k+j*size] = tmp1;
      }
    }else if(k+1 > 32){
      int all = k+1;
      int offset = 0;
      while(all > 32){
        for(int i=0; i<32; i++){
          hipMemcpyAsync((double*)(cu_d200+((i+offset)*size)), (double*)(vmtx+((i+offset)*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream[i]);
          kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream[i]>>>(this->size, cu_d1, (double*)(cu_d200+((i+offset)*size)), (double*)(cu_d201+((i+offset)*BlockPerGrid)));
        }
        hipDeviceSynchronize();
        hipMemcpyAsync(cu_h8, cu_d201, sizeof(double)*BlockPerGrid*(k+1), hipMemcpyDeviceToHost, stream[0]);
        hipDeviceSynchronize();

/* #pragma omp parallel for reduction(+:tmp1) schedule(static) firstprivate(hmtx, eta, cu_h8) lastprivate(hmtx) */
        for(int j=offset; j<offset+32; j++){
          tmp1 = 0.0;
          for(int q=0; q<BlockPerGrid; q++){
            tmp1 += cu_h8[j*BlockPerGrid+q];
          }
          hmtx[k+j*size] = tmp1;
        }
        offset += 32;
        all -= 32;
      }
      for(int i=0; i<all; i++){
        hipMemcpyAsync((double*)(cu_d200+((i+offset)*size)), (double*)(vmtx+((i+offset)*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream[i]);
        kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream[i]>>>(this->size, cu_d1, (double*)(cu_d200+((i+offset)*size)), (double*)(cu_d201+((i+offset)*BlockPerGrid)));
        hipDeviceSynchronize();
        hipMemcpyAsync(cu_h8, cu_d201, sizeof(double)*BlockPerGrid*(k+1), hipMemcpyDeviceToHost, stream[0]);
        hipDeviceSynchronize();

/* #pragma omp parallel for reduction(+:tmp1) schedule(static) firstprivate(hmtx, eta, cu_h8) lastprivate(hmtx) */
        for(int j=offset; j<offset+all; j++){
          tmp1 = 0.0;
          for(int q=0; q<BlockPerGrid; q++){
            tmp1 += cu_h8[j*BlockPerGrid+q];
          }
          hmtx[k+j*size] = tmp1;
        }
      }
    }
  }else{
    //odd
    /* for(int i=0; i<=k; i++){ */
    /*   hmtx[k+i*size] = dot(wvec, vmtx, i, size); */
    /* } */
    if(k <= 32){
      for(int i=0; i<k; i++){
        hipMemcpyAsync((double*)(cu_d200+(i*size)), (double*)(vmtx+(i*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream[i]);
        kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream[i]>>>(this->size, cu_d1, (double*)(cu_d200+(i*size)), (double*)(cu_d201+(i*BlockPerGrid)));
      }
      hipDeviceSynchronize();
      hipMemcpyAsync(cu_h8, cu_d201, sizeof(double)*BlockPerGrid*(k+1), hipMemcpyDeviceToHost, stream[0]);
      hipDeviceSynchronize();

/* #pragma omp parallel for reduction(+:tmp1) schedule(static) firstprivate(hmtx, eta, cu_h8) lastprivate(hmtx) */
      for(int j=0; j<k; j++){
        tmp1 = 0.0;
        for(int q=0; q<BlockPerGrid; q++){
          tmp1 += cu_h8[j*BlockPerGrid+q];
        }
        hmtx[k+j*size] = tmp1;
      }
      hipMemcpyAsync((double*)(cu_d200+(k*size)), (double*)(vmtx+(k*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream[0]);
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream[0]>>>(this->size, cu_d1, (double*)(cu_d200+(k*size)), (double*)(cu_d201+(k*BlockPerGrid)));
      hipDeviceSynchronize();
      hipMemcpyAsync(cu_h8, cu_d201, sizeof(double)*BlockPerGrid*(k+1), hipMemcpyDeviceToHost, stream[0]);
      hipDeviceSynchronize();
      tmp1 = 0.0;
      for(int q=0; q<BlockPerGrid; q++){
        tmp1 += cu_h8[k*BlockPerGrid+q]; 
      }
      hmtx[k+k*size] = tmp1;
    }else if(k>32){
      int all = k;
      int offset = 0;
      while(all > 32){
        for(int i=0; i<32; i++){
          hipMemcpyAsync((double*)(cu_d200+((i+offset)*size)), (double*)(vmtx+((i+offset)*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream[i]);
          kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream[i]>>>(this->size, cu_d1, (double*)(cu_d200+((i+offset)*size)), (double*)(cu_d201+((i+offset)*BlockPerGrid)));
        }
        hipDeviceSynchronize();
        hipMemcpyAsync(cu_h8, cu_d201, sizeof(double)*BlockPerGrid*(k+1), hipMemcpyDeviceToHost, stream[0]);
        hipDeviceSynchronize();

/* #pragma omp parallel for reduction(+:tmp1) schedule(static) firstprivate(hmtx, eta, cu_h8) lastprivate(hmtx) */
        for(int j=offset; j<offset+32; j++){
          tmp1 = 0.0;
          for(int q=0; q<BlockPerGrid; q++){
            tmp1 += cu_h8[j*BlockPerGrid+q];
          }
          hmtx[k+j*size] = tmp1;
        }
        offset += 32;
        all -= 32;
      }
      for(int i=0; i<all; i++){
        hipMemcpyAsync((double*)(cu_d200+((i+offset)*size)), (double*)(vmtx+((i+offset)*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream[i]);
        kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream[i]>>>(this->size, cu_d1, (double*)(cu_d200+((i+offset)*size)), (double*)(cu_d201+((i+offset)*BlockPerGrid)));
        hipDeviceSynchronize();
        hipMemcpyAsync(cu_h8, cu_d201, sizeof(double)*BlockPerGrid*(k+1), hipMemcpyDeviceToHost, stream[0]);
        hipDeviceSynchronize();

/* #pragma omp parallel for reduction(+:tmp1) schedule(static) firstprivate(hmtx, eta, cu_h8) lastprivate(hmtx) */
        for(int j=offset; j<offset+all; j++){
          tmp1 = 0.0;
          for(int q=0; q<BlockPerGrid; q++){
            tmp1 += cu_h8[j*BlockPerGrid+q];
          }
          hmtx[k+j*size] = tmp1;
        }
      }
      hipMemcpyAsync((double*)(cu_d200+(k*size)), (double*)(vmtx+(k*size)), sizeof(double)*size, hipMemcpyHostToDevice, stream[0]);
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock), stream[0]>>>(this->size, cu_d1, (double*)(cu_d200+(k*size)), (double*)(cu_d201+(k*BlockPerGrid)));
      hipDeviceSynchronize();
      hipMemcpyAsync(cu_h8, cu_d201, sizeof(double)*BlockPerGrid*(k+1), hipMemcpyDeviceToHost, stream[0]);
      hipDeviceSynchronize();
      tmp1 = 0.0;
      for(int q=0; q<BlockPerGrid; q++){
        tmp1 += cu_h8[k*BlockPerGrid+q]; 
      }
      hmtx[k+k*size] = tmp1;
    }
  }

  for(int i=0; i<32; i++){
    hipStreamDestroy(stream[i]);
  }

  this->time->end();
  this->time->dot_time += this->time->getTime();
}

void cuda::ShowDevice(){
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  for(int i=0; i<deviceCount; i++){
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, i);
    std::cout << "Device[" << i << "]has compute capability " << deviceProp.major << "." << deviceProp.minor << std::endl;
  }
}

void cuda::HostRegister(void *ptr, int size){

  this->time->start();
  checkCudaErrors( hipHostRegister(ptr, size, hipHostRegisterPortable) );
  this->time->end();
  /* this->time->reg_time += this->time->getTime(); */
}

void cuda::HostUnregister(void *ptr){
  this->time->start();
  checkCudaErrors( hipHostUnregister(ptr) );
  this->time->end();
  /* this->time->unreg_time += this->time->getTime(); */
}

int cuda::GetDeviceNum(){
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  return deviceCount;
}

void cuda::SetSize_Multi(int size1, int size2){
  this->size1 = size1;
  this->size2 = size2;
}

void cuda::EnableP2P(){
  int GPU1ToGPU2 = 0;
  int GPU2ToGPU1 = 0;

  int GPU1 = 0;
  int GPU2 = 1;

  hipDeviceCanAccessPeer(&GPU1ToGPU2, GPU1, GPU2);
  hipDeviceCanAccessPeer(&GPU2ToGPU1, GPU2, GPU1);

  if(GPU2ToGPU1){
    hipSetDevice(GPU2);
    hipDeviceEnablePeerAccess(GPU1, 0);
  }else{
    std::cout << "GPU1 To GPU2 False" << std::endl;
  }

  if(GPU1ToGPU2){
    hipSetDevice(GPU1);
    hipDeviceEnablePeerAccess(GPU2, 0);
  }else{
    std::cout << "GPU1 To GPU2 False" << std::endl;
  }

}

void cuda::MtxVec_mult_Multi(double *in, double *out, double *val1, int *col1, int *ptr1, double *val2, int *col2, int *ptr2){

  hipStream_t GPU1, GPU2;
  /* hipEvent_t eventA; */

  this->time->start();

  int ThreadPerBlock1 = 128;
  int BlockPerGrid1 = (size1-1) / (ThreadPerBlock1/32)+1;

  int ThreadPerBlock2 = 128;
  int BlockPerGrid2 = (size2-1) / (ThreadPerBlock2/32)+1;

  //d1_1 -> in GPU(0)
  //d1_2 -> in GPU(1)

  //d2_1 -> out GPU(0)
  //d2_1 -> out GPU(1)

  checkCudaErrors (hipSetDevice(0));
  checkCudaErrors (hipStreamCreate(&GPU1) );
  /* checkCudaErrors (hipEventCreate(&eventA) ); */
  checkCudaErrors (hipMemsetAsync(cu_d2_1, 0, size1, GPU1) );
  checkCudaErrors (hipMemcpyAsync(cu_d1_1, in, size*sizeof(double), hipMemcpyHostToDevice, GPU1));
  /* checkCudaErrors (hipEventRecord(eventA, GPU1)); */
  kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1, 0, GPU1>>>(this->size1, val1, col1, ptr1, cu_d1_1, cu_d2_1);
  checkCudaErrors( hipPeekAtLastError() );
  checkCudaErrors( hipMemcpyAsync(out, cu_d2_1, size1*sizeof(double), hipMemcpyDeviceToHost, GPU1) );

  checkCudaErrors (hipSetDevice(1));
  checkCudaErrors (hipStreamCreate(&GPU2) );
  checkCudaErrors (hipMemsetAsync(cu_d2_2, 0, size2, GPU2) );
  checkCudaErrors (hipMemcpyAsync(cu_d1_2, in, size*sizeof(double), hipMemcpyHostToDevice, GPU2) );
  /* checkCudaErrors (hipEventSynchronize(eventA)); */
  /* hipMemcpyPeerAsync(cu_d1_2, 1, cu_d1_1, 0, size*sizeof(double), GPU2); */
  kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2, 0, GPU2>>>(this->size2, val2, col2, ptr2, cu_d1_2, cu_d2_2);
  checkCudaErrors( hipPeekAtLastError() );
  checkCudaErrors( hipMemcpyAsync((double*)(out+size1), cu_d2_2, size2*sizeof(double), hipMemcpyDeviceToHost, GPU2) );

  checkCudaErrors (hipSetDevice(0));
  checkCudaErrors( hipStreamSynchronize(GPU1) );
  checkCudaErrors( hipStreamDestroy(GPU1) );

  checkCudaErrors (hipSetDevice(1));
  checkCudaErrors( hipStreamSynchronize(GPU2) );
  checkCudaErrors( hipStreamDestroy(GPU2) );
  checkCudaErrors (hipSetDevice(0));

  this->time->end();
  this->time->mv_time += this->time->getTime();

}

/* void cuda::Kskip_cg_bicg_base_Multi(double *Ar, double *Ap, double *rvec, double *pvec, const int kskip, double *val1, int *col1, int *ptr1, double *val2, int *col2, int *ptr2){ */
/*  */
/*   int ThreadPerBlock1=128; */
/*   int BlockPerGrid1=(size1-1)/(ThreadPerBlock1/32)+1; */
/*  */
/*   int ThreadPerBlock2=128; */
/*   int BlockPerGrid2=(size2-1)/(ThreadPerBlock2/32)+1; */
/*  */
/*   checkCudaErrors( hipSetDevice(0) ); */
/*   checkCudaErrors( hipMemcpy(cu_d1_1, rvec, size*sizeof(double), hipMemcpyHostToDevice) ); */
/*   kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1>>>(this->size1, val1, col1, ptr1, cu_d1_1, cu_d4_1); */
/*   checkCudaErrors( hipPeekAtLastError() ); */
/*   checkCudaErrors( hipMemcpy(Ar, cu_d4_1, size1*sizeof(double), hipMemcpyDeviceToHost) ); */
/*  */
/*   checkCudaErrors( hipSetDevice(1) ); */
/*   checkCudaErrors( hipMemcpy(cu_d1_2, rvec, size*sizeof(double), hipMemcpyHostToDevice) ); */
/*   kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2>>>(this->size2, val2, col2, ptr2, cu_d1_2, (double*)(cu_d4_2+size1)); */
/*   checkCudaErrors( hipPeekAtLastError() ); */
/*   checkCudaErrors( hipMemcpy((double*)(Ar+size1), (double*)(cu_d4_2+size1), size2*sizeof(double), hipMemcpyDeviceToHost) ); */
/*  */
/*   checkCudaErrors( hipSetDevice(0) ); */
/*   checkCudaErrors( hipMemcpy(cu_d1_1, pvec, size*sizeof(double), hipMemcpyHostToDevice) ); */
/*   kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1>>>(this->size1, val1, col1, ptr1, cu_d1_1, cu_d5_1); */
/*   checkCudaErrors( hipPeekAtLastError() ); */
/*   checkCudaErrors( hipMemcpy(Ap, cu_d5_1, size1*sizeof(double), hipMemcpyDeviceToHost) ); */
/*  */
/*   checkCudaErrors( hipSetDevice(1) ); */
/*   checkCudaErrors( hipMemcpy(cu_d1_2, pvec, size*sizeof(double), hipMemcpyHostToDevice) ); */
/*   kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2>>>(this->size2, val2, col2, ptr2, cu_d1_2, (double*)(cu_d5_2+size1)); */
/*   checkCudaErrors( hipPeekAtLastError() ); */
/*   checkCudaErrors( hipMemcpy((double*)(Ap+size1), (double*)(cu_d5_2+size1), size2*sizeof(double), hipMemcpyDeviceToHost) ); */
/*  */
/*  */
/*   for(int i=1; i<2*kskip+2; i++){ */
/*     checkCudaErrors( hipMemcpyPeer((double*)(cu_d5_2+(i-1)*size), 1, (double*)(cu_d5_1+(i-1)*size), 0, size1*sizeof(double)) ); */
/*     checkCudaErrors( hipMemcpyPeer((double*)(cu_d5_1+(i-1)*size+size1), 1, (double*)(cu_d5_2+(i-1)*size+size1), 0, size2*sizeof(double)) ); */
/*  */
/*     checkCudaErrors( hipSetDevice(0) ); */
/*     kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1>>>(this->size1, val1, col1, ptr1, (double*)(cu_d5_1+(i-1)*size), (double*)(cu_d5_1+(i)*size)); */
/*     checkCudaErrors( hipPeekAtLastError() ); */
/*     checkCudaErrors( hipMemcpy((double*)(Ap+(i)*size), (double*)(cu_d5_1+(i)*size), size1*sizeof(double), hipMemcpyDeviceToHost) ); */
/*  */
/*     checkCudaErrors( hipSetDevice(1) ); */
/*     kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2>>>(this->size2, val2, col2, ptr2, (double*)(cu_d5_2+(i-1)*size), (double*)(cu_d5_2+(i)*size+size1)); */
/*     checkCudaErrors( hipPeekAtLastError() ); */
/*     checkCudaErrors( hipMemcpy((double*)(Ap+(i)*size+size1), (double*)(cu_d5_2+(i)*size+size1), size2*sizeof(double), hipMemcpyDeviceToHost) ); */
/*     if(i<2*kskip+1){ */
/*       checkCudaErrors( hipMemcpyPeer((double*)(cu_d4_2+(i-1)*size), 1, (double*)(cu_d4_1+(i-1)*size), 0, size1*sizeof(double)) ); */
/*       checkCudaErrors( hipMemcpyPeer((double*)(cu_d4_1+(i-1)*size+size1), 1, (double*)(cu_d4_2+(i-1)*size+size1), 0, size2*sizeof(double)) ); */
/*  */
/*       checkCudaErrors( hipSetDevice(0) ); */
/*       kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1>>>(this->size1, val1, col1, ptr1, (double*)(cu_d4_1+(i-1)*size), (double*)(cu_d4_1+(i)*size)); */
/*       checkCudaErrors( hipPeekAtLastError() ); */
/*       checkCudaErrors( hipMemcpy((double*)(Ar+(i)*size), (double*)(cu_d4_1+(i)*size), size1*sizeof(double), hipMemcpyDeviceToHost) ); */
/*  */
/*       checkCudaErrors( hipSetDevice(1) ); */
/*       kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2>>>(this->size2, val2, col2, ptr2, (double*)(cu_d4_2+(i-1)*size), (double*)(cu_d4_2+(i)*size+size1)); */
/*       checkCudaErrors( hipPeekAtLastError() ); */
/*       checkCudaErrors( hipMemcpy((double*)(Ar+(i)*size+size1), (double*)(cu_d4_2+(i)*size+size1), size2*sizeof(double), hipMemcpyDeviceToHost) ); */
/*     } */
/*   } */
/*  */
/* } */

void cuda::Kskip_cg_bicg_base_Multi(double *Ar, double *Ap, double *rvec, double *pvec, const int kskip, double *val1, int *col1, int *ptr1, double *val2, int *col2, int *ptr2){

  hipStream_t GPU1_1, GPU1_2;
  hipStream_t GPU2_1, GPU2_2;
  hipEvent_t E1_1, E1_2;
  hipEvent_t E2_1, E2_2;

  int ThreadPerBlock1=128;
  int BlockPerGrid1=(size1-1)/(ThreadPerBlock1/32)+1;

  int ThreadPerBlock2=128;
  int BlockPerGrid2=(size2-1)/(ThreadPerBlock2/32)+1;

  this->time->start();

  checkCudaErrors( hipSetDevice(0) );
  checkCudaErrors( hipStreamCreate(&GPU1_1) );
  checkCudaErrors( hipStreamCreate(&GPU2_1) );
  checkCudaErrors( hipEventCreate(&E1_1) );
  checkCudaErrors( hipEventCreate(&E2_1) );
  checkCudaErrors( hipMemcpyAsync(cu_d1_1, rvec, size*sizeof(double), hipMemcpyHostToDevice, GPU1_1) );
  kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1, 0, GPU1_1>>>(this->size1, val1, col1, ptr1, cu_d1_1, cu_d4_1);
  checkCudaErrors( hipPeekAtLastError() );

  checkCudaErrors( hipSetDevice(1) );
  checkCudaErrors( hipStreamCreate(&GPU1_2) );
  checkCudaErrors( hipStreamCreate(&GPU2_2) );
  checkCudaErrors( hipEventCreate(&E1_2) );
  checkCudaErrors( hipEventCreate(&E2_2) );
  checkCudaErrors( hipMemcpyAsync(cu_d1_2, rvec, size*sizeof(double), hipMemcpyHostToDevice, GPU1_2) );
  kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2, 0, GPU1_2>>>(this->size2, val2, col2, ptr2, cu_d1_2, (double*)(cu_d4_2+size1));
  checkCudaErrors( hipPeekAtLastError() );

//saving memory space 
  /* checkCudaErrors( hipSetDevice(0) ); */
  /* checkCudaErrors( hipMemcpyAsync(cu_d1_1, pvec, size*sizeof(double), hipMemcpyHostToDevice, GPU1_1) ); */
  /* kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1, 0, GPU1_1>>>(this->size1, val1, col1, ptr1, cu_d1_1, cu_d5_1); */
  /* checkCudaErrors( hipPeekAtLastError() ); */
  /* checkCudaErrors( hipEventRecord(E1_1, GPU1_1) ); */

  checkCudaErrors( hipSetDevice(0) );
  checkCudaErrors( hipMemcpyAsync((double*)(cu_d4_1+size1), pvec, size*sizeof(double), hipMemcpyHostToDevice, GPU2_1) );
  kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1, 0, GPU2_1>>>(this->size1, val1, col1, ptr1, (double*)(cu_d4_1+size1), cu_d5_1);
  checkCudaErrors( hipPeekAtLastError() );
  /* checkCudaErrors( hipEventRecord(E1_1, GPU1_1) ); */
  checkCudaErrors( hipEventRecord(E1_1, GPU2_1) );


//saving memory space 
  /* checkCudaErrors( hipSetDevice(1) ); */
  /* checkCudaErrors( hipMemcpyAsync(cu_d1_2, pvec, size*sizeof(double), hipMemcpyHostToDevice, GPU1_2) ); */
  /* kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2, 0, GPU1_2>>>(this->size2, val2, col2, ptr2, cu_d1_2, (double*)(cu_d5_2+size1)); */
  /* checkCudaErrors( hipPeekAtLastError() ); */
  /* checkCudaErrors( hipEventRecord(E1_2, GPU1_2) ); */

  checkCudaErrors( hipSetDevice(1) );
  checkCudaErrors( hipMemcpyAsync((double*)(cu_d4_2+size1+size2), pvec, size*sizeof(double), hipMemcpyHostToDevice, GPU2_2) );
  kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2, 0, GPU2_2>>>(this->size2, val2, col2, ptr2, (double*)(cu_d4_2+size1+size2), (double*)(cu_d5_2+size1));
  checkCudaErrors( hipPeekAtLastError() );
  /* checkCudaErrors( hipEventRecord(E1_2, GPU1_2) ); */
  checkCudaErrors( hipEventRecord(E1_2, GPU2_2) );

  for(int i=1; i<2*kskip+2; i++){
    checkCudaErrors( hipSetDevice(0) );
    checkCudaErrors( hipStreamWaitEvent(GPU1_1, E1_1, 0) );
    checkCudaErrors( hipMemcpyPeerAsync((double*)(cu_d5_2+(i-1)*size), 1, (double*)(cu_d5_1+(i-1)*size), 0, size1*sizeof(double), GPU1_1) );
    checkCudaErrors( hipEventRecord(E2_1, GPU1_1) );

    checkCudaErrors( hipSetDevice(1) );
    checkCudaErrors( hipStreamWaitEvent(GPU1_2, E1_2, 0) );
    checkCudaErrors( hipMemcpyPeerAsync((double*)(cu_d5_1+(i-1)*size+size1), 1, (double*)(cu_d5_2+(i-1)*size+size1), 0, size2*sizeof(double), GPU1_2) );
    checkCudaErrors( hipEventRecord(E2_2, GPU1_2) );

    checkCudaErrors( hipSetDevice(0) );
    checkCudaErrors( hipStreamWaitEvent(GPU2_1, E2_1, 0) );
    checkCudaErrors( hipStreamWaitEvent(GPU2_1, E2_2, 0) );
    checkCudaErrors( hipMemcpyAsync((double*)(Ap+(i-1)*size), (double*)(cu_d5_1+(i-1)*size), size1*sizeof(double), hipMemcpyDeviceToHost, GPU2_1) );
    checkCudaErrors( hipStreamWaitEvent(GPU1_1, E2_1, 0) );
    checkCudaErrors( hipStreamWaitEvent(GPU1_1, E2_2, 0) );
    kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1, 0, GPU1_1>>>(this->size1, val1, col1, ptr1, (double*)(cu_d5_1+(i-1)*size), (double*)(cu_d5_1+(i)*size));
    checkCudaErrors( hipPeekAtLastError() );
    checkCudaErrors( hipEventRecord(E1_1, GPU1_1) );

    checkCudaErrors( hipSetDevice(1) );
    checkCudaErrors( hipStreamWaitEvent(GPU2_2, E2_1, 0) );
    checkCudaErrors( hipStreamWaitEvent(GPU2_2, E2_2, 0) );
    checkCudaErrors( hipMemcpyAsync((double*)(Ap+(i-1)*size+size1), (double*)(cu_d5_2+(i-1)*size+size1), size2*sizeof(double), hipMemcpyDeviceToHost, GPU2_2) );
    checkCudaErrors( hipStreamWaitEvent(GPU1_2, E2_1, 0) );
    checkCudaErrors( hipStreamWaitEvent(GPU1_2, E2_2, 0) );
    kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2, 0, GPU1_2>>>(this->size2, val2, col2, ptr2, (double*)(cu_d5_2+(i-1)*size), (double*)(cu_d5_2+(i)*size+size1));
    checkCudaErrors( hipPeekAtLastError() );
    checkCudaErrors( hipEventRecord(E1_2, GPU1_2) );

    if(i<2*kskip+1){
      checkCudaErrors( hipSetDevice(0) );
      checkCudaErrors( hipStreamWaitEvent(GPU1_1, E1_1, 0) );
      checkCudaErrors( hipMemcpyPeerAsync((double*)(cu_d4_2+(i-1)*size), 1, (double*)(cu_d4_1+(i-1)*size), 0, size1*sizeof(double), GPU1_1) );
      checkCudaErrors( hipEventRecord(E2_1, GPU1_1) );

      checkCudaErrors( hipSetDevice(1) );
      checkCudaErrors( hipStreamWaitEvent(GPU1_2, E1_2, 0) );
      checkCudaErrors( hipMemcpyPeerAsync((double*)(cu_d4_1+(i-1)*size+size1), 1, (double*)(cu_d4_2+(i-1)*size+size1), 0, size2*sizeof(double), GPU1_2) );
      checkCudaErrors( hipEventRecord(E2_2, GPU1_2) );

      checkCudaErrors( hipSetDevice(0) );
      checkCudaErrors( hipStreamWaitEvent(GPU2_1, E2_1, 0) );
      checkCudaErrors( hipStreamWaitEvent(GPU2_1, E2_2, 0) );
      checkCudaErrors( hipMemcpyAsync((double*)(Ar+(i-1)*size), (double*)(cu_d4_1+(i-1)*size), size1*sizeof(double), hipMemcpyDeviceToHost, GPU2_1) );
      checkCudaErrors( hipStreamWaitEvent(GPU1_1, E2_1, 0) );
      checkCudaErrors( hipStreamWaitEvent(GPU1_1, E2_2, 0) );
      kernel_MtxVec_mult<<<BlockPerGrid1, ThreadPerBlock1, 0, GPU1_1>>>(this->size1, val1, col1, ptr1, (double*)(cu_d4_1+(i-1)*size), (double*)(cu_d4_1+(i)*size));
      checkCudaErrors( hipPeekAtLastError() );
      checkCudaErrors( hipEventRecord(E1_1, GPU1_1) );

      checkCudaErrors( hipSetDevice(1) );
      checkCudaErrors( hipStreamWaitEvent(GPU2_2, E2_1, 0) );
      checkCudaErrors( hipStreamWaitEvent(GPU2_2, E2_2, 0) );
      checkCudaErrors( hipMemcpyAsync((double*)(Ar+(i-1)*size+size1), (double*)(cu_d4_2+(i-1)*size+size1), size2*sizeof(double), hipMemcpyDeviceToHost, GPU2_2) );
      checkCudaErrors( hipStreamWaitEvent(GPU1_2, E2_1, 0) );
      checkCudaErrors( hipStreamWaitEvent(GPU1_2, E2_2, 0) );
      kernel_MtxVec_mult<<<BlockPerGrid2, ThreadPerBlock2, 0, GPU1_2>>>(this->size2, val2, col2, ptr2, (double*)(cu_d4_2+(i-1)*size), (double*)(cu_d4_2+(i)*size+size1));
      checkCudaErrors( hipPeekAtLastError() );
      checkCudaErrors( hipEventRecord(E1_2, GPU1_2) );
    }
    checkCudaErrors( hipSetDevice(0) );
    checkCudaErrors( hipStreamSynchronize(GPU1_1) );

    checkCudaErrors( hipSetDevice(1) );
    checkCudaErrors( hipStreamSynchronize(GPU1_2) );
  }

  checkCudaErrors( hipSetDevice(0) );
  checkCudaErrors( hipStreamWaitEvent(GPU1_1, E1_1, 0) );
  checkCudaErrors( hipMemcpyAsync((double*)(Ap+(2*kskip+1)*size), (double*)(cu_d5_1+(2*kskip+1)*size), size1*sizeof(double), hipMemcpyDeviceToHost, GPU1_1) );
  checkCudaErrors( hipStreamWaitEvent(GPU2_1, E2_1, 0) );
  checkCudaErrors( hipMemcpyAsync((double*)(Ar+(2*kskip)*size), (double*)(cu_d4_1+(2*kskip)*size), size1*sizeof(double), hipMemcpyDeviceToHost, GPU2_1) );

  checkCudaErrors( hipSetDevice(1) );
  checkCudaErrors( hipStreamWaitEvent(GPU1_2, E1_2, 0) );
  checkCudaErrors( hipMemcpyAsync((double*)(Ap+(2*kskip+1)*size+size1), (double*)(cu_d5_2+(2*kskip+1)*size+size1), size2*sizeof(double), hipMemcpyDeviceToHost, GPU1_2) );
  checkCudaErrors( hipStreamWaitEvent(GPU2_2, E2_2, 0) );
  checkCudaErrors( hipMemcpyAsync((double*)(Ar+(2*kskip)*size+size1), (double*)(cu_d4_2+(2*kskip)*size+size1), size2*sizeof(double), hipMemcpyDeviceToHost, GPU2_2) );

  checkCudaErrors( hipSetDevice(0) );
  checkCudaErrors( hipStreamSynchronize(GPU1_1) );
  checkCudaErrors( hipStreamSynchronize(GPU2_1) );
  checkCudaErrors( hipStreamDestroy(GPU1_1) );
  checkCudaErrors( hipStreamDestroy(GPU2_1) );
  checkCudaErrors( hipEventDestroy(E1_1) );
  checkCudaErrors( hipEventDestroy(E2_1) );

  checkCudaErrors( hipSetDevice(1) );
  checkCudaErrors( hipStreamSynchronize(GPU1_2) );
  checkCudaErrors( hipStreamSynchronize(GPU2_2) );
  checkCudaErrors( hipStreamDestroy(GPU1_2) );
  checkCudaErrors( hipStreamDestroy(GPU2_2) );
  checkCudaErrors( hipEventDestroy(E1_2) );
  checkCudaErrors( hipEventDestroy(E2_2) );
  checkCudaErrors( hipSetDevice(0) );

  this->time->end();
  this->time->mv_time += this->time->getTime();

}
