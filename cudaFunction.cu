#include "hip/hip_runtime.h"
#include "cudaFunction.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipsparse.h"

#include "color.hpp"

__device__ __inline__ double shfl_xor(double value, int const lane)
{
  return __hiloint2double(__shfl_xor(__double2hiint(value), lane),
      __shfl_xor(__double2loint(value), lane)); 
}

__global__ void kernel_dot (const int N, const double *__restrict__ a, const double *__restrict__ b, double *c)
{
  extern __shared__ double cache[];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;

  double temp = 0;
  while (tid < N) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cacheIndex] = temp;
  __syncthreads ();

  int i = blockDim.x / 2;
  while (i != 0) {
    if (cacheIndex < i) {
      cache[cacheIndex] += cache[cacheIndex+i];
    }
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0) {
    c[blockIdx.x] = cache[0];
  }
}

__global__ void kernel_dot (const int N, const double *__restrict__ a, const int aindex, const int asize, const double *__restrict__ b, double *c, const int cindex, const int csize)
{
  extern __shared__ double cache[];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;

  double temp = 0;
  while (tid < N) {
    temp += a[aindex * asize + tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cacheIndex] = temp;
  __syncthreads ();

  int i = blockDim.x / 2;
  while (i != 0) {
    if (cacheIndex < i) {
      cache[cacheIndex] += cache[cacheIndex+i];
    }
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0) {
    c[cindex * csize + blockIdx.x] = cache[0];
  }
}

/* kernel_MtxVec_mult_old_1<<<BlockPerGrid, ThreadPerBlock>>>(size, val, col, ptr, D_in, D_out); */
__global__ void kernel_MtxVec_mult_old_1(int n, double *val, int *col, int *ptr, double *b, double *c){
  long row=blockDim.x * blockIdx.x + threadIdx.x;
  long int i;
  if(row<n){
    double tmp=0.0;
    long int row_start=ptr[row];
    long int row_end=ptr[row+1];
    for(i=row_start;i<row_end;i++){
      tmp+=val[i]*b[col[i]];
    }
    c[row]=tmp;
  }
}

  /* kernel_MtxVec_mult_old_2<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(this->size, val, col, ptr, cu_d1, cu_d2); */
__global__ void kernel_MtxVec_mult_old_2(int n, const double *val, const int *col, const int *ptr, const double *__restrict__ b, double *c){
  extern __shared__ volatile double vals[];

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    double sum = 0.0;
    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      sum += val[jj] * b[col[jj]];
    }

    vals[threadIdx.x] = sum;
    vals[threadIdx.x] = sum = sum + vals[threadIdx.x + 16];
    vals[threadIdx.x] = sum = sum + vals[threadIdx.x + 8];
    vals[threadIdx.x] = sum = sum + vals[threadIdx.x + 4];
    vals[threadIdx.x] = sum = sum + vals[threadIdx.x + 2];
    sum = sum + vals[threadIdx.x+1];

    if(lane == 0){
      c[row] = sum;
    }
  }
}

/* kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d2); */
__global__ void kernel_MtxVec_mult(int n, const double *val, const int *col, const int *ptr, const double *__restrict__ b, double *c){

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    double sum = 0.0;
    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      sum += val[jj] * b[col[jj]];
    }

    sum += shfl_xor(sum, 16);
    sum += shfl_xor(sum, 8);
    sum += shfl_xor(sum, 4);
    sum += shfl_xor(sum, 2);
    sum += shfl_xor(sum, 1);


    if(lane == 0){
      c[row] = sum;
    }
  }
}

__global__ void kernel_MtxVec_mult(int n, const double *val, const int *col, const int *ptr, const double *__restrict__ b, double *c, const int cindex, const int csize){

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    double sum = 0.0;
    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      sum += val[jj] * b[col[jj]];
    }

    sum += shfl_xor(sum, 16);
    sum += shfl_xor(sum, 8);
    sum += shfl_xor(sum, 4);
    sum += shfl_xor(sum, 2);
    sum += shfl_xor(sum, 1);


    if(lane == 0){
      c[cindex * csize + row] = sum;
    }
  }
}

__global__ void kernel_MtxVec_mult(int n, const double *val, const int *col, const int *ptr, const double *__restrict__ b, const int bindex, const int bsize, double *c, const int cindex, const int csize){

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id/32;
  int lane = thread_id & (32 - 1);

  int row = warp_id;
  if(row<n)
  {
    int row_start = ptr[row];
    int row_end = ptr[row+1];

    double sum = 0.0;
    for(int jj = row_start+lane; jj<row_end; jj+=32)
    { 
      sum += val[jj] * b[bindex * bsize + col[jj]];
    }

    sum += shfl_xor(sum, 16);
    sum += shfl_xor(sum, 8);
    sum += shfl_xor(sum, 4);
    sum += shfl_xor(sum, 2);
    sum += shfl_xor(sum, 1);


    if(lane == 0){
      c[cindex * csize + row] = sum;
    }
  }
}

cuda::cuda(){
  time = new times();

  this->cu_d1 = NULL;
  this->cu_d2 = NULL;

  this->cu_d3 = NULL;
  this->cu_h1 = NULL;

  this->cu_d4 = NULL;
  this->cu_d5 = NULL;
  this->cu_d6 = NULL;
  this->cu_d7 = NULL;
  this->cu_d8 = NULL;
  
  this->cu_d9 = NULL;

  this->cu_h2 = NULL;
  this->cu_h3 = NULL;
  this->cu_h4 = NULL;

  this->cu_h5 = NULL;


  this->dot_copy_time = 0.0;
  this->dot_proc_time = 0.0;
  this->dot_malloc_time = 0.0;
  this->dot_reduce_time = 0.0;

  this->MV_copy_time = 0.0;
  this->MV_proc_time = 0.0;
  this->MV_malloc_time = 0.0;

  this->All_malloc_time = 0.0;

}

cuda::cuda(int size) : cuda::cuda(){
  this->size = size;

  this->time->start();
  int tmp = ceil((double)this->size/(double)128);
  this->cu_d1 = d_Malloc(this->size);
  this->cu_d2 = d_Malloc(this->size);

  this->cu_d3 = d_Malloc(tmp);
  this->cu_h1 = new double [tmp];

  this->time->end();
  this->All_malloc_time += this->time->getTime();
}

cuda::cuda(int size, int k) : cuda::cuda(size){
  this->k = k;
  this->time->start();
  int tmp = ceil((double)this->size/(double)128);

  this->cu_d4 = d_Malloc(this->size * (2*this->k + 1));
  this->cu_d5 = d_Malloc(this->size * (2*this->k + 2));
  this->cu_d6 = d_Malloc(tmp * (2*this->k));
  this->cu_d7 = d_Malloc(tmp * (2*this->k + 1));
  this->cu_d8 = d_Malloc(tmp * (2*this->k + 2));

  this->cu_h2 = new double [tmp * (2*this->k)];
  this->cu_h3 = new double [tmp * (2*this->k + 1)];
  this->cu_h4 = new double [tmp * (2*this->k + 2)];

  this->cu_d9 = d_Malloc(tmp * (2*this->k + 1));
  this->cu_h5 = new double [tmp * (2*this->k + 1)];

  this->time->end();
  this->All_malloc_time += this->time->getTime();
}

cuda::~cuda(){

  Free(cu_d1);
  Free(cu_d2);
  Free(cu_d3);
  Free(cu_d4);
  Free(cu_d5);
  Free(cu_d6);
  Free(cu_d7);
  Free(cu_d8);
  Free(cu_d9);
  delete[] cu_h1;
  delete[] cu_h2;
  delete[] cu_h3;
  delete[] cu_h4;
  delete[] cu_h5;

  delete this->time;
}



void cuda::Free(void* ptr){
  checkCudaErrors(hipFree(ptr));
}

void cuda::FreeHost(void* ptr){
  checkCudaErrors(hipHostFree(ptr));
}

void cuda::H2D(double *from, double *to, int size){
  checkCudaErrors(hipMemcpy(to, from, sizeof(double)*size, hipMemcpyHostToDevice));
}

void cuda::D2H(double *from, double *to, int size){
  checkCudaErrors(hipMemcpy(to, from, sizeof(double)*size, hipMemcpyDeviceToHost));
}

void cuda::H2D(int *from, int *to, int size){
  checkCudaErrors(hipMemcpy(to, from, sizeof(int)*size, hipMemcpyHostToDevice));
}

void cuda::D2H(int *from, int *to, int size){
  checkCudaErrors(hipMemcpy(to, from, sizeof(int)*size, hipMemcpyDeviceToHost));
}

double* cuda::d_Malloc(int size){
  double *ptr = NULL;
  const int s = sizeof(double) * size;
  checkCudaErrors(hipMalloc((void**)&ptr, s));
  return ptr;
}

double* cuda::d_MallocHost(int size){
  double *ptr = NULL;
  const int s = sizeof(double) * size;
  checkCudaErrors(hipHostMalloc((void**)&ptr, s));
  return ptr;
}

int* cuda::i_Malloc(int size){
  int *ptr = NULL;
  const int s = sizeof(int) * size;
  checkCudaErrors(hipMalloc((void**)&ptr, s));
  return ptr;
}

int* cuda::i_MallocHost(int size){
  int *ptr = NULL;
  const int s = sizeof(int) * size;
  checkCudaErrors(hipHostMalloc((void**)&ptr, s));
  return ptr;
}

void cuda::Memset(double *ptr, double val, int size){
  checkCudaErrors(hipMemset(ptr, val, sizeof(double)*size));
}

void cuda::Memset(int *ptr, int val, int size){
  checkCudaErrors(hipMemset(ptr, val, sizeof(int)*size));
}

void cuda::Reset(){
  checkCudaErrors(hipDeviceReset());
}

void cuda::MtxVec_mult(double *in, double *out, int size, double *val, int *col, int *ptr){
  double *D_in = NULL, *D_out = NULL;

  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  this->time->start();
  D_in = d_Malloc(size);
  D_out = d_Malloc(size);
  Memset(D_out, 0, size);
  this->time->end();
  this->MV_malloc_time += this->time->getTime();

  this->time->start();
  H2D(in, D_in, size);
  this->time->end();
  this->MV_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }
  this->time->start();
  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock+16)>>>(size, val, col, ptr, D_in, D_out);
  checkCudaErrors( hipPeekAtLastError() );
  this->time->end();
  this->MV_proc_time += this->time->getTime();

  this->time->start();
  D2H(D_out, out, size);
  this->time->end();
  this->MV_copy_time += this->time->getTime();

  this->time->start();
  Free(D_in);
  Free(D_out);
  this->time->end();
  this->MV_malloc_time += this->time->getTime();
}

void cuda::MtxVec_mult(double *in, double *out, double *val, int *col, int *ptr){

  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  this->time->start();
  Memset(this->cu_d2, 0, size);
  this->time->end();
  this->All_malloc_time += this->time->getTime();

  //d1 -> in
  //d2 -> out
  this->time->start();
  H2D(in, this->cu_d1, size);
  this->time->end();
  this->MV_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }
  this->time->start();
  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d2);
  checkCudaErrors( hipPeekAtLastError() );
  this->time->end();
  this->MV_proc_time += this->time->getTime();

  this->time->start();
  D2H(cu_d2, out, size);
  this->time->end();
  this->MV_copy_time += this->time->getTime();
}

void cuda::MtxVec_mult(double *in, int inindex, int insize, double *out, int outindex, int outsize, double *val, int *col, int *ptr){
  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  this->time->start();
  Memset(this->cu_d2, 0, size);
  this->time->end();
  this->All_malloc_time += this->time->getTime();

  //d1 -> in
  //d2 -> out
  this->time->start();
  H2D((double*)(in+(inindex*insize)), this->cu_d1, size);
  this->time->end();
  this->MV_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }
  this->time->start();
  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d2);
  checkCudaErrors( hipPeekAtLastError() );
  this->time->end();
  this->MV_proc_time += this->time->getTime();

  this->time->start();
  D2H(this->cu_d2, (double*)(out+(outindex*outsize)), size);
  this->time->end();
  this->MV_copy_time += this->time->getTime();

}

void cuda::MtxVec_mult(double *in, int inindex, int insize, double *out, double *val, int *col, int *ptr){
  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  this->time->start();
  Memset(this->cu_d2, 0, size);
  this->time->end();
  this->All_malloc_time += this->time->getTime();

  //d1 -> in
  //d2 -> out
  this->time->start();
  H2D((double*)(in+(inindex*insize)), this->cu_d1, size);
  this->time->end();
  this->MV_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }
  this->time->start();
  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d2);
  checkCudaErrors( hipPeekAtLastError() );
  this->time->end();
  this->MV_proc_time += this->time->getTime();

  this->time->start();
  D2H(this->cu_d2, out, size);
  this->time->end();
  this->MV_copy_time += this->time->getTime();

}

double cuda::dot(double *in1, double *in2, int size){
  double *D_in1=NULL, *D_in2=NULL;
  double *H_out=NULL, *D_out=NULL, sum=0.0;

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();
  D_in1 = d_Malloc(size);
  D_in2 = d_Malloc(size);
  D_out = d_Malloc(BlockPerGrid);
  H_out = new double [BlockPerGrid];
  this->time->end();
  this->dot_malloc_time += this->time->getTime();


  this->time->start();
  H2D(in1, D_in1, size);
  H2D(in2, D_in2, size);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  this->time->start();
  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(size, D_in1, D_in2, D_out);
  checkCudaErrors( hipPeekAtLastError() );
  this->time->end();
  this->dot_proc_time += this->time->getTime();

  this->time->start();
  D2H(D_out, H_out, BlockPerGrid);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  this->time->start();
#pragma omp parallel for schedule(static) reduction(+:sum)
  for(int i=0; i<BlockPerGrid; i++){
    sum += H_out[i];
  }
  this->time->end();
  this->dot_reduce_time += this->time->getTime();


  this->time->start();
  delete[] H_out;
  Free(D_in1);
  Free(D_in2);
  Free(D_out);
  this->time->end();
  this->dot_malloc_time += this->time->getTime();

  return sum;
}

double cuda::dot(double *in1, double *in2){
  double sum=0.0;


  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);


  this->time->start();
  Memset(this->cu_d3, 0, BlockPerGrid);
  this->time->end();
  this->All_malloc_time += this->time->getTime();

  //d_1 -> in1
  //d_2 -> in2
  //d_3 -> out
  this->time->start();
  H2D(in1, this->cu_d1, size);
  H2D(in2, this->cu_d2, size);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  this->time->start();
  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d1, cu_d2, cu_d3);
  checkCudaErrors( hipPeekAtLastError() );
  this->time->end();
  this->dot_proc_time += this->time->getTime();

  //d_3 -> out
  //h_1 -> out(host)
  this->time->start();
  D2H(cu_d3, cu_h1, BlockPerGrid);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  this->time->start();
#pragma omp parallel for schedule(static) reduction(+:sum)
  for(int i=0; i<BlockPerGrid; i++){
    sum += cu_h1[i];
  }
  this->time->end();
  this->dot_reduce_time += this->time->getTime();

  return sum;
}


double cuda::dot(double *in1, int in1index, int in1size, double *in2, int in2index, int in2size){
  double sum=0.0;


  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);


  this->time->start();
  Memset(this->cu_d3, 0, BlockPerGrid);
  this->time->end();
  this->All_malloc_time += this->time->getTime();

  //d_1 -> in1
  //d_2 -> in2
  //d_3 -> out
  this->time->start();
  H2D((double*)(in1+(in1index*in1size)), this->cu_d1, size);
  H2D((double*)(in2+(in2index*in2size)), this->cu_d2, size);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  this->time->start();
  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d1, cu_d2, cu_d3);
  checkCudaErrors( hipPeekAtLastError() );
  this->time->end();
  this->dot_proc_time += this->time->getTime();

  //d_3 -> out
  //h_1 -> out(host)
  this->time->start();
  D2H(cu_d3, cu_h1, BlockPerGrid);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  this->time->start();
#pragma omp parallel for schedule(static) reduction(+:sum)
  for(int i=0; i<BlockPerGrid; i++){
    sum += cu_h1[i];
  }
  this->time->end();
  this->dot_reduce_time += this->time->getTime();

  return sum;

}

double cuda::dot(double *in1, double *in2, int in2index, int in2size){
  double sum=0.0;


  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);


  this->time->start();
  Memset(this->cu_d3, 0, BlockPerGrid);
  this->time->end();
  this->All_malloc_time += this->time->getTime();

  //d_1 -> in1
  //d_2 -> in2
  //d_3 -> out
  this->time->start();
  H2D(in1, this->cu_d1, size);
  H2D((double*)(in2+(in2index*in2size)), this->cu_d2, size);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  this->time->start();
  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d1, cu_d2, cu_d3);
  checkCudaErrors( hipPeekAtLastError() );
  this->time->end();
  this->dot_proc_time += this->time->getTime();

  //d_3 -> out
  //h_1 -> out(host)
  this->time->start();
  D2H(cu_d3, cu_h1, BlockPerGrid);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  this->time->start();
#pragma omp parallel for schedule(static) reduction(+:sum)
  for(int i=0; i<BlockPerGrid; i++){
    sum += cu_h1[i];
  }
  this->time->end();
  this->dot_reduce_time += this->time->getTime();

  return sum;
}

double cuda::dot(double *in1, int in1index, int in1size, double *in2){
  double sum=0.0;


  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);


  this->time->start();
  Memset(this->cu_d3, 0, BlockPerGrid);
  this->time->end();
  this->All_malloc_time += this->time->getTime();

  //d_1 -> in1
  //d_2 -> in2
  //d_3 -> out
  this->time->start();
  H2D((double*)(in1+(in1index*in1size)), this->cu_d2, size);
  H2D(in2, this->cu_d2, size);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  this->time->start();
  kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d1, cu_d2, cu_d3);
  checkCudaErrors( hipPeekAtLastError() );
  this->time->end();
  this->dot_proc_time += this->time->getTime();

  //d_3 -> out
  //h_1 -> out(host)
  this->time->start();
  D2H(cu_d3, cu_h1, BlockPerGrid);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  this->time->start();
#pragma omp parallel for schedule(static) reduction(+:sum)
  for(int i=0; i<BlockPerGrid; i++){
    sum += cu_h1[i];
  }
  this->time->end();
  this->dot_reduce_time += this->time->getTime();

  return sum;
}

void cuda::CSR2CSC(double *dCSRval, int *dCSRcol, int *dCSRptr, double *CSCval, int *CSCrow, int *CSCptr, double *dCSCval, int *dCSCrow, int *dCSCptr, int N, int NNZ){
  hipsparseHandle_t handle=0;
  hipsparseCreate(&handle);

  std::cout << "Transpose Matrix in CUDA.........." << std::flush;
  hipsparseStatus_t status = hipsparseDcsr2csc(handle, N, N, NNZ, dCSRval, dCSRptr, dCSRcol, dCSCval, dCSCrow, dCSCptr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
  std::cout << GREEN << "[○] Done" << RESET << std::endl;

  if(status != HIPSPARSE_STATUS_SUCCESS){
    std::cout << "error in cusparse" << std::endl;
    exit(-1);
  }

  hipMemcpy(CSCval, dCSCval, sizeof(double)*NNZ, hipMemcpyDeviceToHost);
  hipMemcpy(CSCrow, dCSCrow, sizeof(int)*NNZ, hipMemcpyDeviceToHost);
  hipMemcpy(CSCptr, dCSCptr, sizeof(int)*(N+1), hipMemcpyDeviceToHost);

}

void cuda::CSR2CSC(double *CSRval, int *CSRcol, int *CSRptr, double *CSCval, int *CSCrow, int *CSCptr, int N, int NNZ){

  double *dCSRval;
  int *dCSRcol, *dCSRptr;
  double *dCSCval;
  int *dCSCrow, *dCSCptr;
  hipsparseHandle_t handle=0;
  hipsparseCreate(&handle);


  hipMalloc((void**)&dCSRval, sizeof(double)*NNZ);
  hipMalloc((void**)&dCSRcol, sizeof(int)*NNZ);
  hipMalloc((void**)&dCSRptr, sizeof(int)*(N+1));

  hipMalloc((void**)&dCSCval, sizeof(double)*NNZ);
  hipMalloc((void**)&dCSCrow, sizeof(int)*NNZ);
  hipMalloc((void**)&dCSCptr, sizeof(int)*(N+1));

  hipMemcpy(dCSRval, CSRval, sizeof(double)*NNZ, hipMemcpyHostToDevice);
  hipMemcpy(dCSRcol, CSRcol, sizeof(int)*NNZ, hipMemcpyHostToDevice );
  hipMemcpy(dCSRptr, CSRptr, sizeof(int)*(N+1),  hipMemcpyHostToDevice);

  memset(CSCval, 0, sizeof(double)*NNZ);
  memset(CSCrow, 0, sizeof(int)*NNZ);
  memset(CSCptr, 0, sizeof(int)*(N+1));

  hipMemset(dCSCval, 0, sizeof(double)*NNZ);
  hipMemset(dCSCrow, 0, sizeof(int)*NNZ);
  hipMemset(dCSCptr, 0, sizeof(int)*(N+1));

  std::cout << "Transpose Matrix in CUDA.........."<< std::flush;
  hipsparseStatus_t status = hipsparseDcsr2csc(handle, N, N, NNZ, dCSRval, dCSRptr, dCSRcol, dCSCval, dCSCrow, dCSCptr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
  std::cout << GREEN << "[○] Done" << RESET << std::endl;

  if(status != HIPSPARSE_STATUS_SUCCESS){
    std::cout << "error in cusparse" << std::endl;
    exit(-1);
  }

  hipMemcpy(CSCval, dCSCval, sizeof(double)*NNZ, hipMemcpyDeviceToHost);
  hipMemcpy(CSCrow, dCSCrow, sizeof(int)*NNZ, hipMemcpyDeviceToHost);
  hipMemcpy(CSCptr, dCSCptr, sizeof(int)*(N+1), hipMemcpyDeviceToHost);

  hipFree(dCSRval);
  hipFree(dCSRcol);
  hipFree(dCSRptr);

  hipFree(dCSCval);
  hipFree(dCSCrow);
  hipFree(dCSCptr);
}

void cuda::Kskip_cg_bicg_base(double *Ar, double *Ap, double *rvec, double *pvec, const int kskip, double *val, int *col, int *ptr){
  int ThreadPerBlock=128;
  int BlockPerGrid=(size-1)/(ThreadPerBlock/32)+1;

  this->time->start();
  Memset(this->cu_d4, 0, size*(2*k+1));
  Memset(this->cu_d5, 0, size*(2*k+2));
  this->time->end();
  this->All_malloc_time += this->time->getTime();

  //r -> d1
  //p -> d2
  this->time->start();
  H2D(rvec, this->cu_d1, size);
  H2D(pvec, this->cu_d2, size);
  this->time->end();
  this->MV_copy_time += this->time->getTime();

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  // d1(in) --> d4(out)
  // d2(in) --> d5(out)
  this->time->start();

  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d1, cu_d4, 0, this->size);
  checkCudaErrors( hipPeekAtLastError() );

  kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d2, cu_d5, 0, this->size);
  checkCudaErrors( hipPeekAtLastError() );

  for(int i=1; i<2*kskip+2; i++){
    if(i<2*kskip+1){
      kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d4, i-1, this->size, cu_d4, i, this->size);
      checkCudaErrors( hipPeekAtLastError() );
    }
    kernel_MtxVec_mult<<<BlockPerGrid, ThreadPerBlock>>>(this->size, val, col, ptr, cu_d5, i-1, this->size, cu_d5, i, this->size);
    checkCudaErrors( hipPeekAtLastError() );
  }

  this->time->end();
  this->MV_proc_time += this->time->getTime();

  this->time->start();
  D2H(this->cu_d4, Ar, size*(2*kskip+1));
  D2H(this->cu_d5, Ap, size*(2*kskip+2));
  this->time->end();
  this->MV_copy_time += this->time->getTime();
}

void cuda::Kskip_cg_innerProduce(double *delta, double *eta, double *zeta, double *Ar, double *Ap, double *rvec, double *pvec, int kskip, double *val, int *col, int *ptr){

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();
  Memset(this->cu_d6, 0, BlockPerGrid * (2*kskip));
  Memset(this->cu_d7, 0, BlockPerGrid * (2*kskip+1));
  Memset(this->cu_d8, 0, BlockPerGrid * (2*kskip+2));
  this->time->end();
  this->All_malloc_time += this->time->getTime();

  //d1 -> r
  //d2 -> p
  //d4 -> Ar
  //d5 -> Ap

  //d6 -> delta
  //d7 -> eta
  //d8 -> zeta

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  this->time->start();
  for(int i=0; i<2*kskip+2; i++){
    if(i<2*kskip){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d4, i, this->size, cu_d1, cu_d6, i, BlockPerGrid);
    }
    if(i<2*kskip+1){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d5, i, this->size, cu_d1, cu_d7, i, BlockPerGrid);
    }
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d5, i, this->size, cu_d2, cu_d8, i, BlockPerGrid);
  }
  this->time->end();
  this->dot_proc_time += this->time->getTime();


  //d6 -> delta -> h2
  //d7 -> eta -> h3
  //d8 -> zeta -> h4

  this->time->start();
  D2H(cu_d6, cu_h2, BlockPerGrid * (2*kskip));
  D2H(cu_d7, cu_h3, BlockPerGrid * (2*kskip+1));
  D2H(cu_d8, cu_h4, BlockPerGrid * (2*kskip+2));
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  double tmp1 = 0.0;
  double tmp2 = 0.0;
  double tmp3 = 0.0;
  for(int i=0; i<2*kskip+2; i++){
    tmp1 = 0.0;
    tmp2 = 0.0;
    tmp3 = 0.0;
    if(i<2*kskip){
      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h2[i*BlockPerGrid+j];
      }
      delta[i] = tmp1;
    }
    if(i<2*kskip+1){
      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h3[i*BlockPerGrid+j];
      }
      eta[i] = tmp2;
    }
    for(int j=0; j<BlockPerGrid; j++){
      tmp3 += cu_h4[i*BlockPerGrid+j];
    }
    zeta[i] = tmp3;
  }
}

void cuda::Kskip_bicg_innerProduce(double *theta, double *eta, double *rho, double *phi, double *Ar, double *Ap, double *r_vec, double *p_vec, int kskip, double *val, int *col, int *ptr){

  int ThreadPerBlock=128;
  int BlockPerGrid=ceil((double)size/(double)ThreadPerBlock);

  this->time->start();
  Memset(this->cu_d6, 0, BlockPerGrid * (2*kskip));
  Memset(this->cu_d7, 0, BlockPerGrid * (2*kskip+1));
  Memset(this->cu_d9, 0, BlockPerGrid * (2*kskip+1));
  Memset(this->cu_d8, 0, BlockPerGrid * (2*kskip+2));
  this->time->end();
  this->All_malloc_time += this->time->getTime();


  this->time->start();
  H2D(r_vec, cu_d1, this->size);
  H2D(p_vec, cu_d2, this->size);
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  //d1 -> *r
  //d2 -> *p
  //d4 -> Ar
  //d5 -> Ap

  //d6 -> theta
  //d7 -> eta
  //d9 -> rho
  //d8 -> phi

  if(ThreadPerBlock*8 >= 49152){
    std::cout << "Request shared memory size is over max shared memory size in per block !!! Max = 49152 !!! Request = " << ThreadPerBlock*8 << std::endl;
  }

  this->time->start();
  for(int i=0; i<2*kskip+2; i++){
    if(i<2*kskip){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d4, i, this->size, cu_d1, cu_d6, i, BlockPerGrid);
    }
    if(i<2*kskip+1){
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d5, i, this->size, cu_d1, cu_d7, i, BlockPerGrid);
      kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d4, i, this->size, cu_d2, cu_d9, i, BlockPerGrid);
    }
    kernel_dot<<<BlockPerGrid, ThreadPerBlock, sizeof(double)*(ThreadPerBlock)>>>(this->size, cu_d5, i, this->size, cu_d2, cu_d8, i, BlockPerGrid);
  }
  this->time->end();
  this->dot_proc_time += this->time->getTime();

  //d6 -> theta -> h2
  //d7 -> eta -> h3
  //d9 -> rho -> h5
  //d8 -> phi -> h4

  this->time->start();
  D2H(cu_d6, cu_h2, BlockPerGrid * (2*kskip));
  D2H(cu_d7, cu_h3, BlockPerGrid * (2*kskip+1));
  D2H(cu_d9, cu_h5, BlockPerGrid * (2*kskip+1));
  D2H(cu_d8, cu_h4, BlockPerGrid * (2*kskip+2));
  this->time->end();
  this->dot_copy_time += this->time->getTime();

  double tmp1 = 0.0;
  double tmp2 = 0.0;
  double tmp3 = 0.0;
  double tmp4 = 0.0;
#pragma omp parallel for reduction(+:tmp1, tmp2, tmp3, tmp4) schedule(static) firstprivate(theta, eta, rho, phi, cu_h2, cu_h3, cu_h4, cu_h5) lastprivate(theta, eta, rho, phi)
  for(int i=0; i<2*kskip+2; i++){
    tmp1 = 0.0;
    tmp2 = 0.0;
    tmp3 = 0.0;
    tmp4 = 0.0;
    if(i<2*kskip){
      for(int j=0; j<BlockPerGrid; j++){
        tmp1 += cu_h2[i*BlockPerGrid+j];
      }
      theta[i] = tmp1;
    }
    if(i<2*kskip+1){
      for(int j=0; j<BlockPerGrid; j++){
        tmp2 += cu_h3[i*BlockPerGrid+j];
        tmp3 += cu_h5[i*BlockPerGrid+j];
      }
      eta[i] = tmp2;
      rho[i] = tmp3;
    }
    for(int j=0; j<BlockPerGrid; j++){
      tmp4 += cu_h4[i*BlockPerGrid+j];
    }
    phi[i] = tmp4;
  }
}
